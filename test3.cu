
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <iostream>
#define TILE_DIM 16                     // Tile dimension
#define DIMX 1                            
#define DIMY 785
#define DIMZ 31

__global__ void MatMulNoShared(float* A, float* B, float* C, int ARows, int ACols, int BRows, int BCols, int CRows, int CCols) {

    float CValue = 0;

    int Row = blockIdx.y*TILE_DIM + threadIdx.y;
    int Col = blockIdx.x*TILE_DIM + threadIdx.x;

    for (int k = 0; k < (TILE_DIM + ACols - 1)/TILE_DIM; k++) {

        for (int n = 0; n < TILE_DIM; ++n) 
            if ((k*TILE_DIM + n < ACols && Row < ARows) && (k*TILE_DIM + n < BRows && Col < BCols))
                CValue += A[Row*ACols + k*TILE_DIM + n] * B[(k*TILE_DIM + n)*BCols + Col];

    }

    if (Row < CRows && Col < CCols) C[((blockIdx.y * blockDim.y + threadIdx.y)*CCols)+(blockIdx.x*blockDim.x)+threadIdx.x]=CValue;
}

int main() {

    int CCols = DIMZ, CRows=DIMX, ACols=DIMY, ARows=DIMX, BCols=DIMZ, BRows=DIMY;

    dim3 dimBlock(TILE_DIM, TILE_DIM, 1);
    dim3 dimGrid;

    dimGrid.x = (CCols + dimBlock.x - 1)/dimBlock.x;
    dimGrid.y = (CRows + dimBlock.y - 1)/dimBlock.y;

    float *deviceA, *deviceB, *deviceC;

    float* hostA    = (float*)malloc(DIMX*DIMY*sizeof(float));
    float* hostB    = (float*)malloc(DIMY*DIMZ*sizeof(float));
    float* hostC    = (float*)malloc(DIMX*DIMZ*sizeof(float));
    float* hostCp   = (float*)malloc(DIMX*DIMZ*sizeof(float));

    for (int x = 0; x<DIMY; x++)
    {
            hostA[x] = x+1;
        for (int y = 0; y<DIMZ; y++) {
            hostB[x*DIMZ+y] = x+y+1;
        }
   }

    hipMalloc((void **)&deviceA, DIMX*DIMY*sizeof(float));
    hipMalloc((void **)&deviceB, DIMY*DIMZ*sizeof(float));
    hipMalloc((void **)&deviceC, DIMX*DIMZ*sizeof(float));

    hipMemcpy(deviceA, hostA, DIMX*DIMY*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, hostB, DIMY*DIMZ*sizeof(float), hipMemcpyHostToDevice);

    MatMulNoShared<<<dimGrid , dimBlock>>>(deviceA , deviceB , deviceC , ARows , ACols, BRows ,BCols , CRows , CCols);
    hipMemcpy(hostC, deviceC, DIMX*DIMZ*sizeof(float), hipMemcpyDeviceToHost);

std::cout << "A=";
for (int i=0;i<ARows;i++)
{
   for (int j=0;j<ACols;j++)
   {
      std::cout << hostA[i*ACols+j] << " ";
   }
 std::cout << std::endl;
}

std::cout << "B=";
for (int i=0;i<BRows;i++)
{
   for (int j=0;j<BCols;j++)
   {
      std::cout << hostB[i*BCols+j] << " ";
   }
 std::cout << std::endl;
}
std::cout << "C=";
for (int i=0;i<CRows;i++)
{
   for (int j=0;j<CCols;j++)
   {
      std::cout << hostC[i*CCols+j] << " ";
   }
 std::cout << std::endl;
}
    return 0;
}
