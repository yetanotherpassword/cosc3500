#include "hip/hip_runtime.h"
// COSC3500, Semester 2, 2021
// Assignment 2
// Main file - CUDA version
// Built under Visual Studio 2019 (Community Ed) & Nvida Tool Kit 11.4
// And using ARPACK windows prebuilt binaries from http://wo80.bplaced.net/math/packages.html namely ARPACK 3.7.0 and its dependency SUPERLU 5.2.1
// Once all installed/unzipped
// Click START menu on Windows10, select "x64 Native Tools Command Prompt for Visual Studio 2019"
// nvcc --compile eigensolver.cpp -o eigensolver.o
// nvcc --compile randutil.cpp -o randutil.o
// nvcc - O2 --gpu-architecture=sm_35 -Wno-deprecated-gpu-targets Assignment2_cuda.cu eigensolver.o randutil.o -l ..\arpack\arpack-3.7.0-shared\arpack-3.7.0-shared\x64\libarpack -o Assignment2_cuda.exe
#include "eigensolver.h"
#include "randutil.h"
#include <string>
#include <iostream>
#include <iomanip>

//  For goliath (getafix - not required)
//  module load cuda/10.1 gcc

using namespace std;
// global variables to store the matrix

double* M = nullptr;
double* Q = nullptr;
int N = 0;
double* mDevice; // input matrix to multply to 
double* xDevice; // this input vector, which gives
double* yDevice; // this resultant output vector for returning to cpu from gpu

int Threads = 256;
int Blocks;
int Grids;


void checkError(hipError_t e)
{
    if (e != hipSuccess)
    {
        std::cerr << "CUDA error: " << int(e) << " : " << hipGetErrorString(e) << '\n';
        abort();
    }
}

__global__
void CUDA_MatrixVectorMultiply(int n, double* M, double* Y, const double* X)
{


    // blockDim is the number of threads in a block
    // gridDim is the number of blocks in the grid
    int xindex = blockIdx.x * blockDim.x + threadIdx.x;
    int yindex = blockIdx.y * blockDim.y + threadIdx.y;
    int xstride = blockDim.x * gridDim.x;
    int ystride = blockDim.y * gridDim.y;
    for (int i = xindex; i < n; i+= xstride)
    {
        Y[i] = 0;
        for (int j = yindex; j < n; j+= ystride)
        {
             Y[i] += M[i * n + j] * X[j];
        }
    }
}



// implementation of the matrix-vector multiply function
void MatrixVectorMultiply(double* Y, const double* X)
{

    // copy memory from host to device
 
    //checkError(hipMemcpy(xDevice, X, N * sizeof(double), hipMemcpyHostToDevice));

    checkError(hipMemcpy(xDevice, X, N * sizeof(double), hipMemcpyHostToDevice));
    CUDA_MatrixVectorMultiply<<<Grids, Blocks>>> (N, mDevice, xDevice, yDevice);
    checkError(hipDeviceSynchronize());
    checkError(hipMemcpy(Y, yDevice, N * sizeof(double), hipMemcpyDeviceToHost));
    for (int i=0;i<N;i++)
    cout <<"Y["<<i<<"]="<<Y[i]<<endl;


}

int main(int argc, char** argv)
{
   // get the current time, for benchmarking
   auto StartTime = std::chrono::high_resolution_clock::now();

   // get the input size from the command line
   if (argc < 2)
   {
      std::cerr << "expected: matrix size <N>\n";
      return 1;
   }
   N = std::stoi(argv[1]);
   
   Blocks = 32;
   Grids = (N + Blocks - 1) / Threads;

   // Allocate memory for the matrix
   M = static_cast<double*>(malloc(N*N*sizeof(double)));
   Q = static_cast<double*>(malloc(N*N*sizeof(double)));

   // seed the random number generator to a known state
   randutil::seed(4);  // The standard random number.  https://xkcd.com/221/

   // Initialize the matrix.  This is a matrix from a Gaussian Orthogonal Ensemble.
   // The matrix is symmetric.
   // The diagonal entries are gaussian distributed with variance 2.
   // The off-diagonal entries are gaussian distributed with variance 1.
   for (int i = 0; i < N; ++i)
   {
      M[i*N+i] = std::sqrt(2.0) * randutil::randn();
      for (int j = i+1; j < N; ++j)
      {
         M[i*N + j] = M[j*N + i] = randutil::randn();
      }
   }
   // allocate memory on the device

   checkError(hipMalloc(&mDevice, N * N * sizeof(double)));
   checkError(hipMalloc(&xDevice, N * sizeof(double)));
   checkError(hipMalloc(&yDevice, N * sizeof(double)));

   checkError(hipMemcpy(mDevice, M, N * N * sizeof(double), hipMemcpyHostToDevice));
   checkError(hipMemcpy(Q, mDevice, N * N * sizeof(double), hipMemcpyDeviceToHost));
   for (int i = 0; i < N; ++i)
   {
      for (int j = 0; j < N; ++j)
      {
         cout << "M=" << M[i*N + j] << " Q=" <<Q[i*N + j] << endl;
      }
   }
exit(0);
   auto FinishInitialization = std::chrono::high_resolution_clock::now();

   // Call the eigensolver
   EigensolverInfo Info = eigenvalues_arpack(N, 100);

   auto FinishTime = std::chrono::high_resolution_clock::now();

   auto InitializationTime = std::chrono::duration_cast<std::chrono::microseconds>(FinishInitialization - StartTime);
   auto TotalTime = std::chrono::duration_cast<std::chrono::microseconds>(FinishTime - StartTime);

   std::cout << "Obtained " << Info.Eigenvalues.size() << " eigenvalues.\n";
   std::cout << "The largest eigenvalue is: " << std::setw(16) << std::setprecision(12) << Info.Eigenvalues.back() << '\n';
   std::cout << "Total time:                             " << std::setw(12) << TotalTime.count() << " us\n";
   std::cout << "Time spent in initialization:           " << std::setw(12) << InitializationTime.count() << " us\n";
   std::cout << "Time spent in eigensolver:              " << std::setw(12) << Info.TimeInEigensolver.count() << " us\n";
   std::cout << "   Of which the multiply function used: " << std::setw(12) << Info.TimeInMultiply.count() << " us\n";
   std::cout << "   And the eigensolver library used:    " << std::setw(12) << (Info.TimeInEigensolver - Info.TimeInMultiply).count() << " us\n";
   std::cout << "Total serial (initialization + solver): " << std::setw(12) << (TotalTime - Info.TimeInMultiply).count() << " us\n";
   std::cout << "Number of matrix-vector multiplies:     " << std::setw(12) << Info.NumMultiplies << '\n';
   std::cout << "Time per matrix-vector multiplication:  " << std::setw(12) << (Info.TimeInMultiply / Info.NumMultiplies).count() << " us\n";

   // free memory
   free(M);
   checkError(hipFree(mDevice));
   checkError(hipFree(xDevice));
   checkError(hipFree(yDevice));
}
