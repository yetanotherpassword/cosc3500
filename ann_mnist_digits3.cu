#include "hip/hip_runtime.h"
#include <iomanip>
#include <cmath>
#include <chrono>
//#include <boost/algorithm/string.hpp>
//#include <boost/algorithm/string/split.hpp>
#include <vector>
#include <limits>
#include <sstream>
#include <fstream>
#include <iostream>
#include <string>
// Application Parameters
#define DEFTHREADS 256
#define INPUT_LINES 784
#define OUTPUT_LINES 10
#define MATRIX_SIDE 28
#define MAX_PIXEL_VAL 255.0f
#define IMAGE_OFFSET 16
#define DEFAULT_HIDDEN 30
#define ETA_DEFAULT 0.5f
#define EPSILON 1E-04
#define TRAININGSAMPLES 60000
#define TESTINGSAMPLES 10000
#define EPOCHS 1

// How often to print samples, 1=All, 2=every second one, etc
// Undefine or define to very large number to remove output
#define SAMPLEFREQ 1
#undef SAMPLEFREQ



void checkError(hipError_t e)
{
     if (e != hipSuccess)
     {
          std::cerr << "CUDA error: " << int(e) << " : " << hipGetErrorString(e) <<
               '\n';
          abort();
     }
}


/*
 * ALLAN CAMPTON
 * COSC3500 Milestone 2 Parallel Version
 *
 * To perform a full build and run from scratch, do the following
 *
     unzip Project_AC.zip
     cd ~/cosc3500/
     unzip mnist.zip
     make
     sbatch ./goslurm.sh ann_mnist_digits_cuda    #Run parallel version (with default settings)
     sbatch ./goslurm.sh ann_mnist_digits_serial  #Run serial version for comparison 

 */


int thrds = DEFTHREADS;

using namespace std;

float mintime = std::numeric_limits<float>::max();
float maxtime = std::numeric_limits<float>::min();

std::chrono::microseconds Process_MaxTime = std::chrono::microseconds::min();
std::chrono::microseconds Process_MinTime = std::chrono::microseconds::max();
std::chrono::microseconds Call_MaxTime = std::chrono::microseconds::min();
std::chrono::microseconds Call_MinTime = std::chrono::microseconds::max();
std::chrono::microseconds Avg_Time;
int avgcnt=0;

#ifndef SERIAL_ONLY
double *LayerWeightsDevice;
double *ActuationDevice;
double *NetinDevice;
hipEvent_t start, stop;
int tile_dimension = 8; 
#endif

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


#define TILE_DIM 16                     // Tile dimension


class newmat {
public:
   double * ptr;
   int n_rows;
   int n_cols;
   newmat(int r, int c)
   {
       n_rows=r;
       n_cols=c;
       ptr=new double [r*c];
   };
   string prtstr()
   { 
       string s="";
       for (int i=0;i<n_rows;i++)
       {
          for (int j=0;j<n_cols;j++)
		   s+= "   " + to_string(ptr[i*n_cols+j]);
	  s+= '\n';
       }
       return s;
   };
   void free_ele()
   {
       if (ptr != NULL)
          delete [] ptr;
   };
   void zeroize()
   {
       for (int i=0;i<n_rows;i++)
       {
           for (int j=0;j<n_cols;j++)
		  ptr[i*n_cols+j]=0.0;
       }
   };
   double * memptr()
   {
       return ptr;
   };
   int index_max_row(int r, int start, int stop)
   {
        int idx=0;
        double max=  std::numeric_limits<double>::min();
        if (((r<n_rows) && (r>=0)) && (start>=0) && (start < n_cols) && (stop >=0) && (stop<n_cols) && (start<=stop))
          for (int i =r; i<=r;i++)
             for (int j =start; j<=stop;j++)
               if (ptr[i*n_cols+j] > max)
               {
                  idx=i*n_cols+j;
                  max =  ptr[i*n_cols+j];
               }
        return idx;
   };

 } ;
 
__global__ void MatMulNoShared(double* A, double* B, double* C, int ARows, int ACols, int BRows, int BCols, int CRows, int CCols) {

    double CValue = 0;

    int Row = blockIdx.y*TILE_DIM + threadIdx.y;
    int Col = blockIdx.x*TILE_DIM + threadIdx.x;

    for (int k = 0; k < (TILE_DIM + ACols - 1)/TILE_DIM; k++) {

        for (int n = 0; n < TILE_DIM; ++n) 
            if ((k*TILE_DIM + n < ACols && Row < ARows) && (k*TILE_DIM + n < BRows && Col < BCols))
                CValue += A[Row*ACols + k*TILE_DIM + n] * B[(k*TILE_DIM + n)*BCols + Col];

    }

    if (Row < CRows && Col < CCols) C[((blockIdx.y * blockDim.y + threadIdx.y)*CCols)+(blockIdx.x*blockDim.x)+threadIdx.x]=CValue;
}

void PreMatMul(newmat & a, newmat & b, newmat & c)
{
    int DIMZ = c.n_cols;
    int DIMX = c.n_rows;
    int DIMY = a.n_cols;
    if ((DIMX != a.n_rows) || (DIMY != b.n_rows) || (DIMZ != b.n_cols))
    {
       cout << "Incorrect dimensions passed to PreMatMul" << endl;
       exit(1);
    }

    int CCols = DIMZ, CRows=DIMX, ACols=DIMY, ARows=DIMX, BCols=DIMZ, BRows=DIMY;

    dim3 dimBlock(TILE_DIM, TILE_DIM, 1);
    dim3 dimGrid;

    dimGrid.x = (CCols + dimBlock.x - 1)/dimBlock.x;
    dimGrid.y = (CRows + dimBlock.y - 1)/dimBlock.y;
cout << " dimGrid.x = ("<< CCols << " + " << dimBlock.x << " - 1)/" << dimBlock.x<<endl;
cout << " dimGrid.y = ("<< CRows << " + " << dimBlock.y << " - 1)/" << dimBlock.y<<endl;
    double *deviceA, *deviceB, *deviceC;
    //hostC = 
    double* hostC    = (double*)malloc(DIMX*DIMZ*sizeof(double));

    hipMalloc((void **)&deviceA, DIMX*DIMY*sizeof(double));
    hipMalloc((void **)&deviceB, DIMY*DIMZ*sizeof(double));
    hipMalloc((void **)&deviceC, DIMX*DIMZ*sizeof(double));

    hipMemcpy(deviceA, a.memptr(), DIMX*DIMY*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, b.memptr(), DIMY*DIMZ*sizeof(double), hipMemcpyHostToDevice);

    auto StartChronoTime = std::chrono::high_resolution_clock::now();

                   auto StartCallTime = std::chrono::high_resolution_clock::now();

    MatMulNoShared<<<dimGrid , dimBlock>>>(deviceA , deviceB , deviceC , ARows , ACols, BRows ,BCols , CRows , CCols);
	
    checkError(hipDeviceSynchronize());


    auto EndCallTime = std::chrono::high_resolution_clock::now();
    auto TotalCallTime = std::chrono::duration_cast<std::chrono::microseconds > (EndCallTime - StartCallTime);


    if (TotalCallTime > Call_MaxTime)
        Call_MaxTime = TotalCallTime;

    if (TotalCallTime < Call_MinTime)
        Call_MinTime = TotalCallTime;


    hipMemcpy(hostC, deviceC, DIMX*DIMZ*sizeof(double), hipMemcpyDeviceToHost);
    for (int j=0;j<31;j++)
      cout << hostC[j] << " " ;
    cout << endl;

    memcpy(c.memptr(), hostC, DIMX*DIMZ*sizeof(double));

}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////





std::time_t result = std::time(nullptr);
string fid = to_string(result);
unsigned int NumberOfLayers;
unsigned int OutputLayer;
unsigned int *nodes;
double eta;	// Learning factor

vector<newmat> netin;
vector<newmat> actuation;
vector<newmat> deltafn;
vector<newmat> ftick;
vector<newmat> layer_weights;
vector<newmat> weight_updates;
vector<newmat> new_layer_weights;


ios init(NULL);
stringstream confusion_matrix;
newmat err_summary(1,OUTPUT_LINES);


#ifdef WANT_TO_LOAD_WEIGHTS
// Used for loading weights from file (if ever required)
double l2[10][50000];
int nd[100];
int nd2[100];
int lays;
int t = 0;
int x = 0;
#endif

#ifdef SERIAL_ONLY
string build_type = "Serial";
#else
string build_type = "Parallel";
#endif




// implementation of the matrix-vector multiply function
void SerialMatrixVectorMultiply(double *Y, double *X, double *M, int m_nr, int m_nc)
{
  // Need to ensure Y vector passed has been zeroised
    for (int i=0;i<m_nr*m_nc ;i++)
    {
        int c1=i % m_nc;
        int r1=i / m_nc;
        Y[c1] += X[r1] * M[c1 *m_nr + r1];
    }
}

void sigmoid3(newmat & net, newmat & out)
{
   int c=net.n_cols;
   for (int i=0;i<c;i++)
     out.ptr[i] = 1 / (1 + exp(-net.ptr[i]));
   out.ptr[c] = 1.0;	// add bias signal value
     //return out;
}

/////////////////////////////////////////////
//
// PRINT ROUTINES
//
void print_an_image_vals(unsigned char *c, int i)
{
     cout << "This is a : " << i << endl << flush;
     for (int i = 0; i < INPUT_LINES; i++)
     {
          if (i % MATRIX_SIDE == 0)
               cout << endl << flush;
          cout << hex << std::setfill('0') << std::setw(2) << (unsigned int) c[i] <<
               dec << " ";
     }
     cout << endl << flush;
}

void print_an_image(unsigned char *c, int i)
{
     cout << "This is a : " << i << endl << flush;
     for (int i = 0; i < INPUT_LINES; i++)
     {
          if (i % MATRIX_SIDE == 0)
               cout << endl << flush;
          if (c[i] == 0)
               cout << "  ";
          else if (c[i] < 128)
               cout << "xx";
          else
               cout << "XX";
     }
     cout << endl << flush;
}

void print_images(unsigned char *c, int size)
{
     for (int i = IMAGE_OFFSET; i < size; i++)
     {
          if (((i - IMAGE_OFFSET) % MATRIX_SIDE) == 0)
               cout << endl << flush;
          if (((i - IMAGE_OFFSET) % INPUT_LINES) == 0)
               cout << endl << "Image : " << dec <<
               ((i - IMAGE_OFFSET) / INPUT_LINES) + 1 << endl << flush;
          cout << hex << std::setfill('0') << std::setw(2) << (unsigned int) c[i] <<
               " ";
     }
}

//
//
/////////////////////////////////////////////

unsigned char *load_file(string filename, string labels, unsigned char **labs)
{
     unsigned char *memblock;
     ifstream inFile;
     streampos size;

     cout << "Using file '" << filename << "'" << endl << flush;
    	//
    	// Load MNIST DIGIT IMAGES
    	//
     inFile.open(filename, ios:: in | ios::binary | ios::ate);
     if (!inFile)
     {
          cout << "Unable to open file '" << filename << "'" << endl << flush;
          exit(1);	// terminate with error
     }
     else
     {
          cout << "OK opened '" << filename << "' Successfully" << endl << flush;
     }

     if (inFile.is_open())
     {
          size = inFile.tellg();
          memblock = new unsigned char[size];
          inFile.seekg(0, ios::beg);
          inFile.read((char*) memblock, size);
          inFile.close();

          cout << "the entire file content is in memory, all " << size <<
               " bytes of it" << endl << flush;
     }
     inFile.close();
    	//
    	// Load MNIST DIGIT LABELS
    	//
     inFile.open(labels, ios:: in | ios::binary | ios::ate);
     if (!inFile)
     {
          cout << "Unable to open file '" << labels << "'" << endl << flush;
          exit(1);	// terminate with error
     }
     else
     {
          cout << "OK opened '" << labels << "' Successfully" << endl << flush;
     }

     if (inFile.is_open())
     {
          size = inFile.tellg();
          *labs = new unsigned char[size];
          inFile.seekg(0, ios::beg);
          inFile.read((char*) *labs, size);
          inFile.close();

          cout << "the entire file content is in memory, all " << size <<
               " bytes of it" << endl << flush;
     }
     inFile.close();
     return memblock;
}

void load_an_image(int seq, unsigned char* &mptr, newmat &img, newmat &t,
     unsigned char* &lp)
{
     int start = (INPUT_LINES *seq) + IMAGE_OFFSET;
     double greyval = MAX_PIXEL_VAL;

     for (int i = 0; i < INPUT_LINES; i++)
     {
          img.ptr[i] = ((double) mptr[start + i]) / greyval;
     }

     img.ptr[nodes[0]] = 1;      // set bias signal, so can multiply with[node weights |
        // bias weights] augmented matrix

     int img_is_digit = (int) lp[8 + seq];
#ifdef SAMPLEFREQ
     if ((seq + 1) % SAMPLEFREQ == 0)
     {
          cout << "For sample :" << seq + 1 << endl << flush;
          print_an_image(&mptr[start], img_is_digit);
     }
#endif
     t.zeroize();  // create the target vector (plus one for 'bias' bit)
     if (img_is_digit > 9)
     {
          cout << "Error: img_is_digit=" << img_is_digit << "seq=" << seq << endl;
          exit(1);
     }
     t.ptr[img_is_digit] = 1;    // set the target 'bit'
}


////////////////////////
//
// DEBUG ROUTINES
// For use with gdb
/*
void output(mat t)
{
     cout << t << endl;
}

// For use with gdb
void output(rowvec t)
{
     cout << t << endl;
}
*/
void output(newmat t)
{
     cout << t.prtstr();
}
     
double accu(newmat m1)
{
  double tmp=0;
    for (int i=0;i<m1.n_rows;i++)
      for (int j=0; i<m1.n_cols;j++)
         tmp += m1.ptr[i*m1.n_cols+j];
  return tmp;
}    
newmat diff (newmat p1, newmat p2)
{
  newmat tmp(p1.n_rows, p1.n_cols);
  for (int i=0;i<p1.n_rows;i++)
	 {
        for (int j=0;j<p1.n_cols;j++)
		   tmp.ptr[i*p1.n_cols+j]=p1.ptr[i*p1.n_cols+j] - p2.ptr[i*p1.n_cols+j];
	}
	return tmp;
}
newmat piecewisemult (newmat p1, newmat p2)
{
  newmat tmp(p1.n_rows, p1.n_cols);
  for (int i=0;i<p1.n_rows;i++)
	 {
        for (int j=0;j<p1.n_cols;j++)
		   tmp.ptr[i*p1.n_cols+j]=p1.ptr[i*p1.n_cols+j] * p2.ptr[i*p1.n_cols+j];
	}
	return tmp;
}
newmat matmult (newmat p1, newmat p2)
{
  newmat tmp(p1.n_rows, p2.n_cols);
  if (p1.n_cols == p2.n_rows)
  {
     for (int i=0;i<p1.n_rows;i++)
	 {
        for (int j=0;j<p1.n_cols;j++)
		   tmp.ptr[i*p1.n_cols+j]=p1.ptr[i*p1.n_cols+j] * p2.ptr[i*p1.n_cols+j];
	 }
  }
  return tmp;
}
newmat mult (newmat p1, double p2)
{
  newmat tmp(p1.n_rows, p1.n_cols);
  for (int i=0;i<p1.n_rows;i++)
	 {
        for (int j=0;j<p1.n_cols;j++)
		   tmp.ptr[i*p1.n_cols+j]=p1.ptr[i*p1.n_cols+j] * p2;
	}
	return tmp;
}
newmat add (newmat p1, double p2)
{
  newmat tmp(p1.n_rows, p1.n_cols);
  for (int i=0;i<p1.n_rows;i++)
	 {
        for (int j=0;j<p1.n_cols;j++)
		   tmp.ptr[i*p1.n_cols+j]=p1.ptr[i*p1.n_cols+j] + p2;
	}
	return tmp;
}
newmat matadd (newmat p1, newmat p2)
{
  newmat tmp(p1.n_rows, p1.n_cols);
  for (int i=0;i<p1.n_rows;i++)
	 {
        for (int j=0;j<p1.n_cols;j++)
		   tmp.ptr[i*p1.n_cols+j]=p1.ptr[i*p1.n_cols+j] + p2.ptr[i*p1.n_cols+j];
	}
	return tmp;
}
int backprop(newmat & tgt, int y0)
{

     newmat final = actuation[OutputLayer];
     final.n_cols--;
     newmat tgt0 = tgt;
     //tgt0.insert_cols(nodes[OutputLayer], 1);
     double err = accu(piecewisemult(diff(tgt , final) , diff(tgt , final))) *0.5;
     if (abs(err) < EPSILON)
     {
          int val = tgt0.index_max_row(0,0,9);
#ifdef SAMPLEFREQ
          if ((y0 + 1) % SAMPLEFREQ == 0)
               cout << "---------------------------------- BACK PROPAGATION  sample=" <<
               y0 + 1 << " err=" << err << "<epsilon, for tgt '" << val <<
               "' so error is acceptable, returning" << endl << flush;
#endif
          err_summary.ptr[val] = err;
          return 1;
     }

#ifdef SAMPLEFREQ
     if ((y0 + 1) % SAMPLEFREQ == 0)
          cout << "------------------------------------ BACK PROPAGATION sample=" <<
          y0 + 1 << endl << flush;
#endif
     ftick[OutputLayer] = add( mult(actuation[OutputLayer], -1.0)  , 1.0);
     ftick[OutputLayer] =
          piecewisemult(ftick[OutputLayer] , actuation[OutputLayer]);	// element wise multiply
     deltafn[OutputLayer] = piecewisemult(diff(tgt0 , actuation[OutputLayer]) , ftick[OutputLayer]);

     for (int i = OutputLayer - 1; i >= 0; i--)
     {
          weight_updates[i] =  matmult(deltafn[i + 1], actuation[i]);
          new_layer_weights[i]  = matadd( matmult(layer_weights[i], weight_updates[i]),  mult(weight_updates[i], eta) );
          ftick[i] = add(mult(actuation[i] ,(-1.0)) , 1.0);
          ftick[i] = piecewisemult(ftick[i] , actuation[i]);	// element wise multiply
          deltafn[i] = matmult(deltafn[i + 1] ,layer_weights[i]);
          deltafn[i] = piecewisemult(deltafn[i] , ftick[i]);
     }
     for (int i = 0; i < OutputLayer; i++)
     {

          layer_weights[i] = new_layer_weights[i];
     }
     return 0;
}


void forward_feed(unsigned char* &imgdata, unsigned char* &labdata, bool train,
     int samples)
{
     newmat tgt(1,OUTPUT_LINES+1);
     int tot_correct = 0;
     int tot_wrong = 0;
     int correct_num = -1;
     int best_guess = -1;
     int num_correct[OUTPUT_LINES] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };
     int num_wrong[OUTPUT_LINES] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };
     int chosen_wrongly[OUTPUT_LINES][OUTPUT_LINES] = {
		{ 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },
          { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },
          { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },
          { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },
          { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },
          { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },
          { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },
          { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },
          { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },
          { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 }
     };
     int num_tested = 0;
     int epochs;
     string intype;
     if (train)
     {
          intype = "TRAINING";
          epochs = EPOCHS;
     }
     else
     {
          epochs = 1;
          intype = "TEST    ";
     }
     for (int y = 0; y < samples; y++)
     {
#ifdef SAMPLEFREQ
          if ((y + 1) % SAMPLEFREQ == 0)
               cout << "------------------------------------ FORWARD FEED OF " << intype <<
               " SAMPLE # " << y + 1 << endl << flush;
#endif
          load_an_image(y, imgdata, actuation[0], tgt, labdata);

          int tgtval = tgt.index_max_row(0, 0, 9);
          for (int e = 0; e < epochs; e++)
          {
               for (int i = 0; i < OutputLayer; i++)	// only n-1 transitions between n layers
               {
#ifdef SERIAL_ONLY
                   auto StartCallTime = std::chrono::high_resolution_clock::now();



                   SerialMatrixVectorMultiply(netin[i].ptr, 
                                               actuation[i].ptr, 
                                               layer_weights[i].ptr,  layer_weights[i].n_rows,  layer_weights[i].n_cols);

                   auto EndCallTime = std::chrono::high_resolution_clock::now();
                   auto TotalCallTime = std::chrono::duration_cast<std::chrono::microseconds > 
                                                                       (EndCallTime - StartCallTime);

                   if (TotalCallTime > Call_MaxTime)
                      Call_MaxTime = TotalCallTime;

                   if (TotalCallTime < Call_MinTime)
                      Call_MinTime = TotalCallTime;

                   Avg_Time += TotalCallTime;
                   avgcnt++;

                    for (int j = 0; j < netin[i].n_cols; j++)
                    {
                          netin[i].ptr[j] /= actuation[i].n_cols;
                    }
#ifdef SAMPLEFREQ
                    if ((y + 1) % SAMPLEFREQ == 0)
                         cout << "Netin serial (" << netin[i].n_rows << "," << netin[i].n_cols <<
                         ")= " << netin[i].prtstr("NetIn Serial") << endl << flush;
#endif
#else
                   PreMatMul(actuation[i], layer_weights[i], netin[i]);

#ifdef SAMPLEFREQ
                         cout << "Netin Parallel " << netin[i].n_rows << "," << netin[i].n_cols <<
                         ")= " << netin[i].prtstr("Net In Parallel") << endl << flush;
#endif

#endif
                    sigmoid3(netin[i], actuation[i + 1]);
               }
#ifdef SAMPLEFREQ
               if ((y + 1) % SAMPLEFREQ == 0)
               {
                    std::cout << "Final output : " << endl << std::setw(7) << fixed <<
                         showpoint << actuation[OutputLayer].prtstr("Final Out") <<
                         " Sample: " << y + 1 << std::endl << flush;
                    std::cout << "Expec output : " << endl << std::setw(7) << fixed <<
                         showpoint << tgt.prtstr("Tgt3") << " Sample: " << y + 1 <<
                         std::endl << flush;
               }
#endif
              	//////////////////////////// forward feed end
               if (train)
               {
                   	// printout intermediate result
               //     int outval = actuation[OutputLayer].subvec(0, 9).index_max();
                    int outval =  actuation[OutputLayer].index_max_row(0, 0, 9);
#ifdef SAMPLEFREQ
                    if ((y + 1) % SAMPLEFREQ == 0)
                    {
                         std::cout << "Train output : " << endl << std::setw(7) << fixed <<
                              showpoint << actuation[OutputLayer].prtstr("") <<
                              " Sample: " << y + 1 << std::endl << flush;
                        	// Below just figures out the order in which to print the "A"ctal
                        	// result and "O"bjective result
                        	// (or "*" if correct) in the output line.
                        	// So tgtval is correct if lastval==firstval(they are indicies, and
                        	// will be equal if tgtval==outval)
                         int firstval = tgtval < outval ? tgtval : outval;
                         int lastval = tgtval > outval ? tgtval : outval;
                         string firststr = tgtval == firstval ?
                              to_string(firstval) + string("T") :
                              to_string(firstval) + string("O");
                         string laststr = tgtval == lastval ? to_string(lastval) + "T" :
                              to_string(lastval) + "O";
                         if (firstval == lastval)
                              firststr = "*" + to_string(firstval);	// correct
                         for (int z1 = 0; z1 < firstval; z1++)
                              cout << "         ";
                         cout << "       " << firststr;
                         for (int z1 = 0; z1 < lastval - firstval - 1; z1++)
                              cout << "         ";
                         if (firstval != lastval)
                              cout << "       " << laststr;	// expected
                         cout << endl << flush;
                    }
#endif
                    if (backprop(tgt, y) == 1)
                         break;	// exit i/epoch loop and goto next sample (as error function is
                   	// within limits for this tgt)
               }
          }

          if (!train)
          {
               correct_num = tgt.index_max_row(0, 0, 9);
               best_guess = actuation[OutputLayer].index_max_row(0, 0, 9);

               if (best_guess == correct_num)
               {
                    num_correct[correct_num]++;
                    tot_correct++;
               }
               else
               {
                    num_wrong[correct_num]++;
                    chosen_wrongly[correct_num][best_guess]++;
                    tot_wrong++;
               }
               num_tested++;
          }
          if (!train)
          {
               std::cout << "Final output : " << endl << std::setw(7) << fixed <<
                    showpoint << actuation[OutputLayer].prtstr() <<
                    " Sample: " << y + 1 << std::endl << flush;
               for (int z1 = 0; z1 < actuation[OutputLayer].index_max_row(0, 0, 9); z1++)
                    cout << "         ";
               cout << "       ^" << endl << flush;
               std::cout << "Expec output : " << endl << std::setw(7) << fixed <<
                    showpoint << tgt.prtstr() << " Sample: " << y + 1 <<
                    std::endl << flush;
          }
     }
     if (!train)
     {
          confusion_matrix << "Tested         " << num_tested << " samples" << endl <<
               flush;
          confusion_matrix << "Tested Correct " << tot_correct << " samples" << endl <<
               flush;
          confusion_matrix << "Tested Wrong   " << tot_wrong << " samples" << endl <<
               endl << endl << "  " << flush;
          for (int i = 0; i < OUTPUT_LINES; i++)
               confusion_matrix << "     " << dec << std::setw(6) << "'" << i << "'";
          confusion_matrix << "<-- ANN chose" << endl << flush;
          confusion_matrix << "------------------------------------------------------"
          "------------------------------------------------------"
          "-----------------------------";
          double colsum[OUTPUT_LINES] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };
          double rowsum[OUTPUT_LINES] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };
          string blanks = "                    ";
          for (int i = 0; i < OUTPUT_LINES; i++)
          {
               string correct_size = to_string(num_correct[i]);
               confusion_matrix << endl << setw(4) << i << "  |";
               for (int j = 0; j < OUTPUT_LINES; j++)
               {
                    rowsum[i] += chosen_wrongly[i][j];
                    colsum[j] += chosen_wrongly[i][j];
                    if (i == j)
                         confusion_matrix << std::setw(6) << "[" << num_correct[i] << "]" <<
                         blanks.substr(0, 5 - correct_size.length()) <<
                         "|";
                    else
                         confusion_matrix << std::setw(7) << chosen_wrongly[i][j] << "     |";
               }
               float pctg = 0;
               if (tot_wrong != 0)
                    pctg = (float)(rowsum[i]) / (float)(tot_wrong) *100.0f;
               confusion_matrix << "  " << setw(7) << std::setw(7);
               confusion_matrix.copyfmt(init);
               confusion_matrix << rowsum[i];
               confusion_matrix << setw(7) << "     " << fixed << showpoint << pctg <<
                    "%" << endl << flush;
               confusion_matrix.copyfmt(init);
               confusion_matrix << "----------------------------------------------------"
               "----------------------------------------------------"
               "---------------------------------";
          }
          confusion_matrix << endl << "   ^   ";
          for (int i = 0; i < OUTPUT_LINES; i++)
               confusion_matrix << dec << std::setw(7) << colsum[i] << "      ";
          confusion_matrix << endl << "Target   ";
          for (int i = 0; i < OUTPUT_LINES; i++)
          {
               float pctg = 0;
               if (tot_wrong != 0)
                    pctg = (float)(colsum[i]) / (float)(tot_wrong) *100.0f;
               confusion_matrix << dec << setw(7) << fixed << showpoint << pctg <<
                    "%     ";
               confusion_matrix.copyfmt(init);
          }
          confusion_matrix << endl << endl << endl << endl << endl <<
               "Correct selections:" << endl << flush;
          confusion_matrix << "       ";
          for (int i = 0; i < OUTPUT_LINES; i++)
               confusion_matrix << dec << std::setw(6) << "'" << i << "'     ";
          confusion_matrix << endl << "       ";
          for (int i = 0; i < OUTPUT_LINES; i++)
          {
               confusion_matrix << std::setw(7) << num_correct[i] << "      ";
          }
          confusion_matrix << endl << endl << "Incorrect selections:" << endl <<
               flush;
          confusion_matrix << "       ";
          for (int i = 0; i < OUTPUT_LINES; i++)
               confusion_matrix << dec << std::setw(6) << "'" << i << "'     ";
          confusion_matrix << endl << "       ";
          for (int i = 0; i < OUTPUT_LINES; i++)
          {
               confusion_matrix << std::setw(7) << num_wrong[i] << "      ";
          }
          confusion_matrix << endl << endl << flush;
          float pctg =
               (float)(tot_correct) / (float)(tot_correct + tot_wrong) *100.0f;
          confusion_matrix << "Total Correct : " << std::setw(7) << fixed << showpoint <<
               pctg << "%     " << endl << endl << flush;
          cout << confusion_matrix.str() << flush;
          confusion_matrix.copyfmt(init);
          cout.copyfmt(init);
     }
}

#ifdef WANT_TO_LOAD_WEIGHTS
void load_weights(string fname)
{
     ifstream iFile;
     cout << "Loading weights from file : " << fname << endl << flush;
     iFile.open(fname, ios:: in);
     string aline;

     vector<string> strs;

     if (fname.substr(0, 4) == "post")
          stringstream confusion_matrix2;
     getline(iFile, aline);
     boost::split(strs, aline, boost::is_any_of("="));
     if (strs.size() > 1)
          lays = stoi(strs[1]);
     cout << " Has " << lays << "layers" << endl;
     while (iFile.good())
     {
          getline(iFile, aline);
          if (aline.find("NodesInLayer") != std::string::npos)
          {
               nd2[t] = x;
               x = 0;
               t++;

               boost::split(strs, aline, boost::is_any_of("="));
               if (strs.size() > 1)
                    nd[t] = stoi(strs[1]);
               cout << " Has " << nd[t] << "layers" << endl;
          }
          else if ((aline.find("Error Summary") != std::string::npos))
          {
               nd2[t] = x;
               x = 0;
               t++;
               break;
          }
          else if (aline.find("LayerBiases") == std::string::npos)
          {
               boost::trim(aline);
               boost::split(strs, aline, boost::is_any_of(" "));
               boost::algorithm::split(strs, aline, boost::is_any_of("\t "),
                    boost::token_compress_on);
               for (int y = 0; y < strs.size(); y++)
               {
                    if (strs[y].length() > 0)
                    {
                         l2[t][x++] = stod(strs[y]);
                    }
               }
          }
     }
     for (int i = 0; i < lays; i++)
     {
          for (int j = 0; j < nd2[i + 1]; j++)
          {
               int r = (j) / (nd[i + 1] + 1);
               int c = (j) % (nd[i + 1] + 1);
               layer_weights[i](r, c) = l2[i + 1][j];
          }
     }
}
#endif

void save_weights(string hdr)
{
     ofstream oFile;
     string fname = hdr + string("_weights_") + fid + string(".txt");
     cout << "Saving weights to file : " << fname << endl << flush;
     oFile.open(fname, ios::out);
     if (hdr.substr(0, 4) == "post")
          oFile << confusion_matrix.str();
     oFile << "NumberOfLayers=" << NumberOfLayers << endl << flush;
     for (int i = 0; i < OutputLayer; i++)
     {

          oFile << "NodesInLayer" << i << "=" << nodes[i] << endl << setprecision(20) << fixed << showpoint << flush;
          oFile << "    " << layer_weights[i].prtstr();
          oFile << endl;
          // ALTERNATIVELY
          //for (int j=0; j<layer_weights[i].n_rows ;j++)
          //{
          //   for (int k=0; k<layer_weights[i].n_cols; k++)
          //        oFile << setprecision(20) <<  fixed << showpoint << layer_weights[i](j,k) << " " <<flush;
          //    oFile << endl;
          // }
     }
     oFile << "Error Summary" << endl << flush;

     oFile << err_summary.prtstr() << endl << flush;

     oFile << "EndFile" << endl << flush;
     oFile.close();
     cout.copyfmt(init);

}

int main(int argc, char *argv[])
{
     extern char **environ;
     string hname = "";
     //string y="initial_random_values_weights_11337071.txt";
     string y = "initial_random_values_weights_1636260202.txt";
    auto StartChronoTime = std::chrono::high_resolution_clock::now();

     vector<string> strs;
     string bldver = string(__DATE__) + " at time " + string(__TIME__);
     cout << "--------------------------------  Build done on " << bldver << endl <<
          flush;
     init.copyfmt(cout);
     for (int i=0;i<err_summary.n_cols;i++)
        err_summary.ptr[i] = -1.0;
     if (argc < 2)
     {
          NumberOfLayers = 3;
          nodes = new unsigned int[NumberOfLayers];
          nodes[0] = INPUT_LINES;
          nodes[1] = DEFAULT_HIDDEN;
          nodes[2] = OUTPUT_LINES;
          eta = ETA_DEFAULT;
          cout << "Using default setting of \"" << nodes[0] << " " << nodes[1] << " " <<
               nodes[2] << "\" " << endl << flush;
          cout << "And ETA=" << eta << endl << flush;;
     }
     else if (argc < 6)
     {
          cout << "Usage: " << argv[0] << " ETA IN H1[H2 H3 ...] OUT THREADS" << endl <<
               flush;
          cout << "       Where ETA is the learning factor, &" << endl << flush;
          cout
               <<
               "       Where number of parameters after ETA is the number of layers" <<
               endl << flush;
          cout << "       Must have a minimum of 3, i.e. IN H1 OUT" << endl << flush;
          cout << "       And the parameters themselves are numbers, " << endl <<
               flush;
          cout << "       indicating the number of nodes in that layer." << endl <<
               flush;
          cout << "       e.g. \"" << argv[0] << " " << ETA_DEFAULT << " " <<
               INPUT_LINES << " " << DEFAULT_HIDDEN << " " << OUTPUT_LINES << " " <<
               DEFTHREADS << "\" " << endl << flush;
          cout << "       and is the default, if no params supplied." << endl <<
               flush;
          exit(1);
     }
     else
     {
          NumberOfLayers = argc - 3;
          nodes = new unsigned int[NumberOfLayers];
          eta = stod(string(argv[1]));
          if (eta <= 0)
          {
               cout << "Error: ETA must be positive, usually less than 1" << endl <<
                    flush;
               exit(1);
          }
          for (int i = 2; i < argc - 1; i++)
          {
               int p = stoi(string(argv[i]));
               if (p > 0)
               {
                    nodes[i - 2] = stoi(string(argv[i]));
               }
               else
               {
                    cout << "Error in parameter " << i << " - must be positive" << endl <<
                         flush;
                    exit(1);
               }
          }
          thrds = stoi(argv[argc - 1]);
     }
     cout << "Threads chosen is " << thrds << endl << flush;
     cout << "Number of Layers is " << NumberOfLayers << endl << flush;
/*
    	// netptrs = new double *[NumberOfLayers];
    	// Use slurm job number if avaiable (else defaults to epoch time) for file ids
    	// created
     for (char **current = environ; *current; current++)
     {
          string tmp = *current;
          boost::split(strs, tmp, boost::is_any_of("="));
          if ((strs[0] == "SLURM_JOBID") || (strs[0] == "SLURM_JOB_ID"))
          {
               if (strs[1].length() > 0)
               {
                    fid = strs[1];
               }
          }
          else if (strs[0] == "HOSTNAME")
          {
               if (strs[1].length() > 0)
               {
                    hname = strs[1];
               }
          }
     }
	 */

#ifndef SERIAL_ONLY
 // set up CUDA timing structs
     hipEventCreate(&start);
     hipEventCreate(&stop);
#endif

     OutputLayer = NumberOfLayers - 1;
     unsigned char *trainlabels;
     unsigned char *testlabels;
     unsigned char *traindata = load_file("train-images-idx3-ubyte",
          "train-labels-idx1-ubyte", &trainlabels);
     unsigned char *testdata = load_file("t10k-images-idx3-ubyte",
          "t10k-labels-idx1-ubyte", &testlabels);
     auto StartTime = std::chrono::high_resolution_clock::now();

    	///////////////////////////////////////////////
    	//
    	//  CREATE ARRAY OF MATRICES AND VECTORS
    	//  AND SET WEIGHTS TO RANDOM (0<w < 1)
    	//
     int max_mat = 0;
     int max_vec = 0;
     int bias_field = 1;

     for (int i = 0; i <= OutputLayer; i++)
     {
          max_vec = max(max_vec, (nodes[i] + bias_field));
          newmat rb3a(1, nodes[i] + bias_field);
          actuation.push_back(rb3a);

          newmat drb3(1, nodes[i] + bias_field);
          deltafn.push_back(drb3);

          newmat frb3(1, nodes[i] + bias_field);
          ftick.push_back(frb3);

          if (i < OutputLayer)
          {
               max_mat =
                    max(max_mat, (nodes[i] + bias_field) *(nodes[i + 1] + bias_field));
               // These buffers for the rowvec and mat structures below are done to ensure
               // the Armadillo matrix can be accessed directly and the library doesnt move
               // the memory around
               newmat rb3(1,( nodes[i+1] + bias_field));

               // Create an array of matrices (one element for each layer) for the netin value
               // This holds the sum of weighted signals, for each node, that gets squashed to 
               // produce the nodes output for next layer
               netin.push_back(rb3);
               // Create a buffer of required size for weights, in each layer
               // (plus two more, one for delta updates, and one for holding new weight to be
               // applied after backprop. These maybe consolidated later
               newmat tmpwgt3((nodes[i + 1] + bias_field),( nodes[i] + bias_field));
               for (int p=0;p<(nodes[i + 1] + bias_field)*( nodes[i] + bias_field);p++)
                  tmpwgt3.ptr[p] = rand()/RAND_MAX;
               newmat tmpwgt30((nodes[i + 1] + bias_field),( nodes[i] + bias_field));
               newmat tmpwgt300((nodes[i + 1] + bias_field),( nodes[i] + bias_field));
               // create an array of three matrices (weights for forward prop)
               // and deltas and new values, for back propagation
               layer_weights.push_back(tmpwgt3);

               new_layer_weights.push_back(tmpwgt30);

               weight_updates.push_back(tmpwgt300);
          }
     }
     // Save initial starting weights if required for later
     save_weights("initial_random_values");

    // Informational, the max value of matrix and vectors are record and used to reserve CUDA memory 
     cout << "Max Matrix size " << max_mat << " Max vector size = " << max_vec <<
          endl << flush;

#ifdef WANT_TO_LOAD_WEIGHTS
     // this is a function to load previously saved weights, to either ensure constant initial values
     // if say moving platforms with different psudeo RNG, or to load post weights after training
     // This works, but only implemented atm, by direct code changes, no UI implemented
     // But note used in this project anyway
     load_weights(y);
#endif

#ifndef SERIAL_ONLY
 cout << "CudaMalloc1" << endl << flush;
     checkError(hipMalloc(&ActuationDevice, max_vec* sizeof(double)));
	 cout << "CudaMalloc2" << endl << flush;
     checkError(hipMalloc(&NetinDevice, max_vec* sizeof(double)));
	 cout << "CudaMalloc3" << endl << flush;
     checkError(hipMalloc(&LayerWeightsDevice, max_mat* sizeof(double)));
	 cout << "CudaMalloc4" << endl << flush;
#ifdef __CUDA_ARCH__
     cout << "Built for CUDA ARCH == " << __CUDA_ARCH__ << endl;
#endif
#endif
    	///////////////////////////////////////////////
    	//
    	// TRAIN THE DATA
    	//
     auto StartTrainTime = std::chrono::high_resolution_clock::now();
     cout << "Training on data started (epochs=" << EPOCHS << ")...." << endl <<
          flush;

     forward_feed(traindata, trainlabels, true, TRAININGSAMPLES);
     auto EndTrainTime = std::chrono::high_resolution_clock::now();

     cout << "Training complete" << endl << flush;
    	///////////////////////////////////////////////
    	//
    	// TEST THE DATA
    	//
     cout << "Testing of data started...." << endl << flush;
     auto StartTestTime = std::chrono::high_resolution_clock::now();

     forward_feed(testdata, testlabels, false, TESTINGSAMPLES);

     auto EndTestTime = std::chrono::high_resolution_clock::now();

     cout << "Testing complete" << endl << flush;

     auto TotalTime = std::chrono::duration_cast<std::chrono::microseconds > (          EndTestTime - StartTime);
     auto TrainTime = std::chrono::duration_cast<std::chrono::microseconds > (          EndTrainTime - StartTrainTime);
     auto TestTime = std::chrono::duration_cast<std::chrono::microseconds > (          EndTestTime - StartTestTime);

     cout << "Total Time       : " << std::setw(12) << TotalTime.count() << " us" <<
          endl << flush;
     cout << "Total Train Time : " << std::setw(12) << TrainTime.count() << " us" <<
          endl << flush;
     cout << "Total Test Time  : " << std::setw(12) << TestTime.count() << " us" <<
          endl << flush;

     confusion_matrix << "Epochs in Training : " << EPOCHS << endl << flush;
     confusion_matrix << "Training Samples   : " << TRAININGSAMPLES << endl <<
          flush;
     confusion_matrix << "Testing Samples    : " << TESTINGSAMPLES << endl <<
          flush;
     confusion_matrix << endl << endl << "Total Time       : " << std::setw(12) <<
          TotalTime.count() << " us" << endl << flush;
     confusion_matrix << "Total Train Time : " << std::setw(12) <<
          TrainTime.count() << " us" << endl << flush;
     confusion_matrix << "Total Test Time  : " << std::setw(12) << TestTime.count() <<
          " us" << endl << flush;
     confusion_matrix << endl << endl << "Total Time       : " << std::setw(12) <<
          TotalTime.count() / 1000000 << " s" << endl << flush;
     confusion_matrix << "Total Train Time : " << std::setw(12) <<
          TrainTime.count() / 1000000 << " s" << endl << flush;
     confusion_matrix << "Total Test Time  : " << std::setw(12) <<
          TestTime.count() / 1000000 << " s" << endl << flush;
     confusion_matrix << endl << endl << "Total Time       : " << std::setw(12) <<
          TotalTime.count() / 60000000 << " min" << endl << flush;
     confusion_matrix << "Total Train Time : " << std::setw(12) <<
          TrainTime.count() / 60000000 << " min" << endl << flush;
     confusion_matrix << "Total Test Time  : " << std::setw(12) <<
          TestTime.count() / 60000000 << " min" << endl << flush;
     confusion_matrix << "Epsilon  : " << EPSILON << endl << flush;
     confusion_matrix << "Eta      : " << eta << endl << flush;
     confusion_matrix << "Build ver: " << bldver << endl << flush;
     save_weights("post_training_weights");

     delete[] traindata;
     delete[] trainlabels;
     delete[] testdata;
     delete[] testlabels;

     cout << "Min time for " <<  build_type << " call : " << Call_MinTime.count() << " us" << endl;
     cout << "Min time for " <<  build_type << " call : " << Call_MinTime.count()/1000000  << " s" << endl;
     cout << "Min time for " <<  build_type << " call : " << Call_MinTime.count()/60000000  << " min" << endl;
     cout << "Avg time for " <<  build_type << " call : " << (double) Avg_Time.count()/(double) avgcnt << " us" << endl;
     cout << "Avg time for " <<  build_type << " call : " << (double) Avg_Time.count()/(double) (avgcnt *1000000)  << " s" << endl;
     cout << "Avg time for " <<  build_type << " call : " << (double) Avg_Time.count()/(double) (avgcnt * 60000000)  << " min" << endl;
     cout << "Max time for " <<  build_type << " call : " << Call_MaxTime.count() << " us" << endl;
     cout << "Max time for " <<  build_type << " call : " << Call_MaxTime.count()/1000000  << " s" << endl;
     cout << "Max time for " <<  build_type << " call : " << Call_MaxTime.count()/60000000  << " min" << endl;
     auto EndChronoTime = std::chrono::high_resolution_clock::now();
     auto TotalChronoTime = std::chrono::duration_cast<std::chrono::microseconds > (EndChronoTime - StartChronoTime);

     cout << "Time for  Total Program : " << TotalChronoTime.count() << " us " << endl;
     cout << "Used Tile Dimension of " << tile_dimension << endl;

     for (int i=0;i<netin.size();i++)
         netin[i].free_ele();


#ifndef SERIAL_ONLY
     checkError(hipFree(LayerWeightsDevice));
     checkError(hipFree(ActuationDevice));
     checkError(hipFree(NetinDevice));

     checkError(hipEventDestroy(start));
     checkError(hipEventDestroy(stop));
#endif


}

