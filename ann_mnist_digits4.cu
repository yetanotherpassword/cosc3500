#include "hip/hip_runtime.h"
#include <iomanip>
#include <cmath>
#include <chrono>
#ifdef WANT_TO_LOAD_WEIGHTS
#include <boost/algorithm/string.hpp>
#include <boost/algorithm/string/split.hpp>
#endif
#include <vector>
#include <limits>
#include <sstream>
#include <fstream>
#include <iostream>
#include <string>
// Application Parameters
#define DEFTHREADS 256
#define INPUT_LINES 784
#define OUTPUT_LINES 10
#define MATRIX_SIDE 28
#define MAX_PIXEL_VAL 255.0f
#define IMAGE_OFFSET 16
#define DEFAULT_HIDDEN 30
#define ETA_DEFAULT 0.5f
#define EPSILON 1E-04
#define TRAININGSAMPLES 60000
#define TESTINGSAMPLES 10000
#define EPOCHS 1

// How often to print samples, 1=All, 2=every second one, etc
// Undefine or define to very large number to remove output
#define SAMPLEFREQ 1
//#undef SAMPLEFREQ



void checkError(hipError_t e)
{
    if (e != hipSuccess)
    {
        std::cerr << "CUDA error: " << int(e) << " : " << hipGetErrorString(e) <<
            '\n';
        abort();
    }
}


/*
 * ALLAN CAMPTON
 * COSC3500 Milestone 2 Parallel Version
 *
 * To perform a full build and run from scratch, do the following
 *
     unzip Project_AC.zip
     cd ~/cosc3500/
     unzip mnist.zip
     make
     sbatch ./goslurm.sh ann_mnist_digits_cuda    #Run parallel version (with default settings)
     sbatch ./goslurm.sh ann_mnist_digits_serial  #Run serial version for comparison

 */


int thrds = DEFTHREADS;

using namespace std;

float mintime = std::numeric_limits<float>::max();
float maxtime = std::numeric_limits<float>::min();

std::chrono::microseconds Process_MaxTime = std::chrono::microseconds::min();
std::chrono::microseconds Process_MinTime = std::chrono::microseconds::max();
std::chrono::microseconds Call_MaxTime = std::chrono::microseconds::min();
std::chrono::microseconds Call_MinTime = std::chrono::microseconds::max();
std::chrono::microseconds Avg_Time;
int avgcnt = 0;
char ss[600000];

#ifndef SERIAL_ONLY
double* LayerWeightsDevice;
double* ActuationDevice;
double* NetinDevice;
double* deviceA, * deviceB, * deviceC;
hipEvent_t start, stop;
int tile_dimension = 8;
class newmat;
void PreMatMul(newmat& a, newmat& b, newmat& c, int norm);

#endif

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


#define TILE_DIM 16                     // Tile dimension

#ifndef SERIAL_ONLY 
__global__ void MatMultMat(double* A, double* B, double* C, int ARows, int ACols, int BRows, int BCols, int CRows, int CCols) {

    double CValue = 0;

    int Row = blockIdx.y * TILE_DIM + threadIdx.y;
    int Col = blockIdx.x * TILE_DIM + threadIdx.x;

    for (int k = 0; k < (TILE_DIM + ACols - 1) / TILE_DIM; k++) {

        for (int n = 0; n < TILE_DIM; ++n)
            if ((k * TILE_DIM + n < ACols && Row < ARows) && (k * TILE_DIM + n < BRows && Col < BCols))
                CValue += A[Row * ACols + k * TILE_DIM + n] * B[(k * TILE_DIM + n) * BCols + Col];

    }

    if (Row < CRows && Col < CCols) C[((blockIdx.y * blockDim.y + threadIdx.y) * CCols) + (blockIdx.x * blockDim.x) + threadIdx.x] = CValue;
}

__global__ void TransposeMat(double* A, double* C, int ARows, int ACols) {

    int Row = blockIdx.y * TILE_DIM + threadIdx.y;
    int Col = blockIdx.x * TILE_DIM + threadIdx.x;

    if (Row < ARows && Col < ACols)
        C[Col * ARows + Row] = A[Row * ACols + Col];

}
__global__ void MatMultMatEleWise(double* A, double* B, double* C)
{
    int i = threadIdx.x;
    C[i] = A[i] * B[i];
}


__global__ void MatSubMat(double* A, double* B, double* C)
{
    int i = threadIdx.x;
    C[i] = A[i] - B[i];
}

__global__ void MatAddMat(double* A, double* B, double* C)
{
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}

__global__ void MatAddScalar(double scalar, double* C)
{
    int i = threadIdx.x;
    C[i] = C[i] + scalar;
}

__global__ void MatMultScalar(double scalar, double* C)
{
    int i = threadIdx.x;
    C[i] = C[i] * scalar;
}


#endif
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


class newmat {
public:
    double* ptr;
    int n_rows;
    int n_cols;
    newmat(int r, int c, double* p)
    {
        n_rows = r;
        n_cols = c;
        ptr = new double[n_rows * n_cols];
#ifdef SERIAL_ONLY
        for (int i = 0; i < n_rows; i++)
            for (int j = 0; j < n_cols; j++)
                ptr[i * n_cols + j] = p[i * n_cols + j];
#else
        memcpy(ptr, p, n_rows * n_cols * sizeof(double));
#endif
    };
    newmat()
    {
        n_rows = 0;
        n_cols = 0;
        ptr = NULL;
    };
    newmat(int r, int c)
    {
        n_rows = r;
        n_cols = c;
        ptr = new double[r * c];
    };
    void set_transpose(newmat tmp)
    {
        if (n_rows == tmp.n_cols && n_cols == tmp.n_rows)
        {
#ifdef SERIAL_ONLY
            for (int i = 0; i < tmp.n_rows; i++)
                for (int j = 0; j < tmp.n_cols; j++)
                    ptr[j * tmp.n_rows + i] = tmp.ptr[i * tmp.n_cols + j];
#else
            int onedLen = n_rows * n_cols;
            hipMemcpy(deviceA, tmp.ptr, onedLen * sizeof(double), hipMemcpyHostToDevice);

            TransposeMat <<< 1, onedLen >>> (deviceA, deviceC, tmp.n_rows, tmp.n_cols);


            checkError(hipDeviceSynchronize());

            hipMemcpy(ptr, deviceC, onedLen * sizeof(double), hipMemcpyDeviceToHost);
#endif
        }
        else
        {
            cout << "Cant transpose Matrix[" << tmp.n_rows << "," << tmp.n_cols << "] into given Matrix[" << n_rows << "," << n_cols << "]" << endl << flush;
            exit(1);
        }
    };

    void add_mat(newmat m1)
    {
        if (m1.n_rows != n_rows || m1.n_cols != n_cols)
        {
            cout << "Cant addmat m1[" << m1.n_rows << "," << m1.n_cols << "] with *this[" << n_rows << "," << n_cols << "]" << endl << flush;
            exit(1);
        }
#ifdef SERIAL_ONLY
        for (int i = 0; i < n_rows; i++)
            for (int j = 0; j < n_cols; j++)
                ptr[i * n_cols + j] += m1.ptr[i * n_cols + j];
#else
        int onedLen = n_rows * n_cols;
        hipMemcpy(deviceA, ptr, onedLen * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(deviceB, m1.ptr, onedLen * sizeof(double), hipMemcpyHostToDevice);

        MatAddMat <<< 1, onedLen >>> (deviceA, deviceB, deviceC);

        checkError(hipDeviceSynchronize());

        hipMemcpy(ptr, deviceC, onedLen * sizeof(double), hipMemcpyDeviceToHost);
#endif
    };
    void add_scalar(double d)
    {

#ifdef SERIAL_ONLY
 for (int i = 0; i < n_rows; i++)
     for (int j = 0; j < n_cols; j++)
         ptr[i * n_cols + j] += d;
#else
        int onedLen = n_rows * n_cols;
        hipMemcpy(deviceC, ptr, onedLen * sizeof(double), hipMemcpyHostToDevice);

        MatAddScalar <<< 1, onedLen >>> (d, deviceC);

        checkError(hipDeviceSynchronize());

        hipMemcpy(ptr, deviceC, onedLen * sizeof(double), hipMemcpyDeviceToHost);
#endif
    };
    void mult_scalar(double d)
    {

#ifdef SERIAL_ONLY
for (int i = 0; i < n_rows; i++)
    for (int j = 0; j < n_cols; j++)
        ptr[i * n_cols + j] *= d; 
#else
        int onedLen = n_rows * n_cols;
        hipMemcpy(deviceC, ptr, onedLen * sizeof(double), hipMemcpyHostToDevice);
        
        MatMultScalar <<< 1, onedLen >>> (d, deviceC);

        checkError(hipDeviceSynchronize());

        hipMemcpy(ptr, deviceC, onedLen * sizeof(double), hipMemcpyDeviceToHost);
#endif

    };

    void set_mult1_add2(newmat y1, double d1, double d2)
    {
        if (n_rows != y1.n_rows || n_cols != y1.n_cols)
        {
            cout << "Cant store y1[" << y1.n_rows << "," << y1.n_cols << " in *this[" << n_rows << "," << n_cols << "]" << endl;
            exit(1);
        }

#ifdef SERIAL_ONLY        
        for (int i = 0; i < n_rows; i++)
            for (int j = 0; j < n_cols; j++)
                ptr[i * n_cols + j] = y1.ptr[i * n_cols + j];
#else
        memcpy(ptr, y1.ptr, n_cols * n_rows * sizeof(double));
#endif

        mult_scalar(d1);
        add_scalar(d2);
    };
    void set_mult1_add2(newmat y1, double d1, newmat y2)
    {
        if (n_rows != y1.n_rows || n_rows != y2.n_rows || n_cols != y1.n_cols || n_cols != y2.n_cols)
        {
            cout << "Cant add y1[" << y1.n_rows << "," << y1.n_cols << "] to y2[" << y2.n_rows << "," << y2.n_cols << "] and store in *this[" << n_rows << "," << n_cols << "]" << endl;
            exit(1);
        }
#ifdef SERIAL_ONLY
        for (int i = 0; i < n_rows; i++)
            for (int j = 0; j < n_cols; j++)
                ptr[i * n_cols + j] = y1.ptr[i * n_cols + j];
#else
         memcpy(ptr, y1.ptr, n_cols * n_rows * sizeof(double));
#endif
        mult_scalar(d1);
        add_mat(y2);

    };
    void set_matmult(newmat p1, newmat p2)
    {
        if (p1.n_cols == p2.n_rows)
        {
            if (n_rows != p1.n_rows || n_cols != p2.n_cols)
            {
                cout << "Resultant matrix wont hold result, fixing by realloc in matlmult" << endl;
                free_ele();
                n_rows = p1.n_rows;
                n_cols = p2.n_cols;
                ptr = new double[n_rows * n_cols];
            }
   
#ifdef SERIAL_ONLY
            for (int i = 0; i < p1.n_rows; ++i)
                for (int j = 0; j < p2.n_cols; ++j)
                {
                    ptr[i * p2.n_cols + j] = 0;
                    for (int k = 0; k < p1.n_cols; ++k) // c1==r2
                    {
                        ptr[i * p2.n_cols + j] += p1.ptr[i * p1.n_cols + k] * p2.ptr[k * p2.n_cols + j];
                    }
                }
               
#else
             PreMatMul(p1, p2, *this, 1);
#endif
        }
        else
        {
            cout << "Cant multiply p1[" << p1.n_rows << "," << p1.n_cols << "] by p2[" << p2.n_rows << "," << p2.n_cols << "]" << endl;
            exit(1);
        }
    };

    void piecewisemult(newmat p1)
    {

        if (p1.n_rows != n_rows || p1.n_cols != n_cols)
        {
            cout << "Cant piecewisemultiply p1[" << p1.n_rows << "," << p1.n_cols << "] with *this[" << n_rows << "," << n_cols << "]" << endl;
            exit(1);
        }
#ifdef SERIAL_ONLY
        for (int i = 0; i < p1.n_rows; i++)
        {
            for (int j = 0; j < n_cols; j++)
                ptr[i * n_cols + j] = p1.ptr[i * n_cols + j] * ptr[i * n_cols + j];
        }
#else
        int onedLen = n_rows * n_cols;
        hipMemcpy(deviceA, ptr, onedLen * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(deviceB, p1.ptr, onedLen * sizeof(double), hipMemcpyHostToDevice);

        MatMultMatEleWise <<< 1, onedLen >>> (deviceA, deviceB, deviceC);

        checkError(hipDeviceSynchronize());

        hipMemcpy(ptr, deviceC, onedLen * sizeof(double), hipMemcpyDeviceToHost);
        
#endif
    };

    void set_diff2_piecewisemult3(newmat p1, newmat p2, newmat p3)
    {

        if (n_rows != p1.n_rows || n_rows != p2.n_rows || n_cols != p1.n_cols || n_cols != p2.n_cols)
        {
            cout << "Cant diff p2[" << p2.n_rows << "," << p2.n_cols << "] from p1[" << p1.n_rows << "," << p1.n_cols << "] and store in *this[" << n_rows << "," << n_cols << "]" << endl;
            exit(1);
        }
#ifdef SERIAL_ONLY
        for (int i = 0; i < n_rows; i++)
        {
            for (int j = 0; j < n_cols; j++)
                ptr[i * p1.n_cols + j] = p1.ptr[i * p1.n_cols + j] - p2.ptr[i * p1.n_cols + j];
        }
#else
        int onedLen = n_rows * n_cols;
        hipMemcpy(deviceA, p1.ptr, onedLen * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(deviceB, p2.ptr, onedLen * sizeof(double), hipMemcpyHostToDevice);

        MatSubMat <<< 1, onedLen >>> (deviceA, deviceB, deviceC);

        checkError(hipDeviceSynchronize());

        hipMemcpy(ptr, deviceC, onedLen * sizeof(double), hipMemcpyDeviceToHost);

#endif
        piecewisemult(p3);
    };

    char* prtstr()
    {
        // string s="";
       //  char ss[100000];
        ss[0] = '\0';
        for (int i = 0; i < n_rows; i++)
        {
            for (int j = 0; j < n_cols; j++)
            {
                int len = strlen(ss);
                sprintf(&ss[len], "   %20.10g", ptr[i * n_cols + j]);
                //	   s = s + "   " + to_string(ptr[i*n_cols+j]);
            }
            //  s+= '\n';
            sprintf(&ss[strlen(ss)], "\n");
        }
        return ss;
    };
    void free_ele()
    {
        if (ptr != NULL)
            delete[] ptr;
    };
    void zeroize()
    {
#ifdef SERIAL_ONLY
        for (int i = 0; i < n_rows; i++)
        {
            for (int j = 0; j < n_cols; j++)
                ptr[i * n_cols + j] = 0.0;
        }
#else
        memset(ptr, 0, n_cols * n_rows * sizeof(double));
#endif
    };
    double* memptr()
    {
        return ptr;
    };
    int index_max_row(int r, int start, int stop)
    {
        int idx = 0;
        double max = std::numeric_limits<double>::min();
        if (((r < n_rows) && (r >= 0)) && (start >= 0) && (start < n_cols) && (stop >= 0) && (stop < n_cols) && (start <= stop))
            for (int i = r; i <= r; i++)
                for (int j = start; j <= stop; j++)
                    if (ptr[i * n_cols + j] > max)
                    {
                        idx = i * n_cols + j;
                        max = ptr[i * n_cols + j];
                    }
        return idx;
    };

};

#ifndef SERIAL_ONLY
void PreMatMul(newmat& a, newmat& b, newmat& c, int norm)
{
    int DIMZ = c.n_cols;
    int DIMX = c.n_rows;
    int DIMY = a.n_cols;
    if ((DIMX != a.n_rows) || (DIMY != b.n_rows) || (DIMZ != b.n_cols))
    {
        cout << "Incorrect dimensions passed to PreMatMul" << endl;
        cout << "c(" << c.n_rows << "," << c.n_cols << ") is to be set to "
            << "a(" << a.n_rows << "," << a.n_cols << ") * "
            << "b(" << b.n_rows << "," << b.n_cols << ")  "
            << endl;
        exit(1);
    }

    int CCols = DIMZ, CRows = DIMX, ACols = DIMY, ARows = DIMX, BCols = DIMZ, BRows = DIMY;

    dim3 dimBlock(TILE_DIM, TILE_DIM, 1);
    dim3 dimGrid;

    dimGrid.x = (CCols + dimBlock.x - 1) / dimBlock.x;
    dimGrid.y = (CRows + dimBlock.y - 1) / dimBlock.y;
    //cout << " dimGrid.x = ("<< CCols << " + " << dimBlock.x << " - 1)/" << dimBlock.x<<endl;
    //cout << " dimGrid.y = ("<< CRows << " + " << dimBlock.y << " - 1)/" << dimBlock.y<<endl;
        //hostC = 
    double* hostC = (double*)malloc(DIMX * DIMZ * sizeof(double));

    hipMemcpy(deviceA, a.memptr(), DIMX * DIMY * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, b.memptr(), DIMY * DIMZ * sizeof(double), hipMemcpyHostToDevice);

    auto StartChronoTime = std::chrono::high_resolution_clock::now();

    auto StartCallTime = std::chrono::high_resolution_clock::now();

    MatMultMat << <dimGrid, dimBlock >> > (deviceA, deviceB, deviceC, ARows, ACols, BRows, BCols, CRows, CCols);

    checkError(hipDeviceSynchronize());


    auto EndCallTime = std::chrono::high_resolution_clock::now();
    auto TotalCallTime = std::chrono::duration_cast<std::chrono::microseconds> (EndCallTime - StartCallTime);


    if (TotalCallTime > Call_MaxTime)
        Call_MaxTime = TotalCallTime;

    if (TotalCallTime < Call_MinTime)
        Call_MinTime = TotalCallTime;


    hipMemcpy(hostC, deviceC, DIMX * DIMZ * sizeof(double), hipMemcpyDeviceToHost);
    if (norm != 1)
    {
        for (int i = 0; i < c.n_rows; i++)
        {
            for (int j = 0; j < c.n_cols; j++)
            {
                hostC[i * c.n_cols + j] /= (double)norm;
                cout << hostC[i * c.n_cols + j] << " ";
            }
            cout << endl;
        }
    }

    memcpy(c.memptr(), hostC, DIMX * DIMZ * sizeof(double));

}
#endif
std::time_t result = std::time(nullptr);
string fid = to_string(result);
unsigned int NumberOfLayers;
unsigned int OutputLayer;
unsigned int* nodes;
double eta;	// Learning factor

vector<newmat> netin;
vector<newmat> actuation;
vector<newmat> deltafn;
vector<newmat> deltafn_t;
vector<newmat> ftick;
vector<newmat> layer_weights;
vector<newmat> layer_weights_t;
vector<newmat> weight_updates;
vector<newmat> new_layer_weights;
newmat tgt(1, OUTPUT_LINES + 1);


ios init(NULL);
stringstream confusion_matrix;
newmat err_summary(1, OUTPUT_LINES);


#ifdef WANT_TO_LOAD_WEIGHTS
// Used for loading weights from file (if ever required)
double l2[10][50000];
int nd[100];
int nd2[100];
int lays;
int t = 0;
int x = 0;
#endif

#ifdef SERIAL_ONLY
string build_type = "Serial";
#else
string build_type = "Parallel";
#endif




// implementation of the matrix-vector multiply function
void SerialMatrixVectorMultiply(double* Y, double* X, int r1, double* M, int m_nr, int m_nc)
{
    // Need to ensure Y vector passed has been zeroised


    for (int i = 0; i < r1; ++i)
    {
        for (int j = 0; j < m_nc; ++j)
        {
            Y[i * m_nc + j] = 0;
            for (int k = 0; k < m_nr; ++k) // c1==r2
            {
                Y[i * m_nc + j] += X[i * m_nr + k] * M[k * m_nc + j];
                //                cout << "Y["<<i*m_nc+j<<"] += "<<X[i*m_nr+k] * M[k*m_nc+j] << endl;
            }
        }
    }

}

void sigmoid3(newmat& net, newmat& out)
{
    int c = net.n_cols - 1;
    for (int i = 0; i <= c; i++)
        out.ptr[i] = 1 / (1 + exp(-net.ptr[i]));
    out.ptr[c] = 1.0;	// add bias signal value
      //return out;
}

/////////////////////////////////////////////
//
// PRINT ROUTINES
//
void print_an_image_vals(unsigned char* c, int i)
{
    cout << "This is a : " << i << endl << flush;
    for (int i = 0; i < INPUT_LINES; i++)
    {
        if (i % MATRIX_SIDE == 0)
            cout << endl << flush;
        cout << hex << std::setfill('0') << std::setw(2) << (unsigned int)c[i] <<
            dec << " ";
    }
    cout << endl << flush;
}

void print_an_image(unsigned char* c, int i)
{
    cout << "This is a : " << i << endl << flush;
    for (int i = 0; i < INPUT_LINES; i++)
    {
        if (i % MATRIX_SIDE == 0)
            cout << endl << flush;
        if (c[i] == 0)
            cout << "  ";
        else if (c[i] < 128)
            cout << "xx";
        else
            cout << "XX";
    }
    cout << endl << flush;
}

void print_images(unsigned char* c, int size)
{
    for (int i = IMAGE_OFFSET; i < size; i++)
    {
        if (((i - IMAGE_OFFSET) % MATRIX_SIDE) == 0)
            cout << endl << flush;
        if (((i - IMAGE_OFFSET) % INPUT_LINES) == 0)
            cout << endl << "Image : " << dec <<
            ((i - IMAGE_OFFSET) / INPUT_LINES) + 1 << endl << flush;
        cout << hex << std::setfill('0') << std::setw(2) << (unsigned int)c[i] <<
            " ";
    }
}

//
//
/////////////////////////////////////////////

unsigned char* load_file(string filename, string labels, unsigned char** labs)
{
    unsigned char* memblock;
    ifstream inFile;
    streampos size;

    cout << "Using file '" << filename << "'" << endl << flush;
    //
    // Load MNIST DIGIT IMAGES
    //
    inFile.open(filename, ios::in | ios::binary | ios::ate);
    if (!inFile)
    {
        cout << "Unable to open file '" << filename << "'" << endl << flush;
        exit(1);	// terminate with error
    }
    else
    {
        cout << "OK opened '" << filename << "' Successfully" << endl << flush;
    }

    if (inFile.is_open())
    {
        size = inFile.tellg();
        memblock = new unsigned char[size];
        inFile.seekg(0, ios::beg);
        inFile.read((char*)memblock, size);
        inFile.close();

        cout << "the entire file content is in memory, all " << size <<
            " bytes of it" << endl << flush;
    }
    inFile.close();
    //
    // Load MNIST DIGIT LABELS
    //
    inFile.open(labels, ios::in | ios::binary | ios::ate);
    if (!inFile)
    {
        cout << "Unable to open file '" << labels << "'" << endl << flush;
        exit(1);	// terminate with error
    }
    else
    {
        cout << "OK opened '" << labels << "' Successfully" << endl << flush;
    }

    if (inFile.is_open())
    {
        size = inFile.tellg();
        *labs = new unsigned char[size];
        inFile.seekg(0, ios::beg);
        inFile.read((char*)*labs, size);
        inFile.close();

        cout << "the entire file content is in memory, all " << size <<
            " bytes of it" << endl << flush;
    }
    inFile.close();
    return memblock;
}

void load_an_image(int seq, unsigned char*& mptr, newmat& img, newmat& t,
    unsigned char*& lp)
{
    int start = (INPUT_LINES * seq) + IMAGE_OFFSET;
    double greyval = MAX_PIXEL_VAL;

    for (int i = 0; i < INPUT_LINES; i++)
    {
        img.ptr[i] = ((double)mptr[start + i]) / greyval;
    }

    img.ptr[nodes[0]] = 1;      // set bias signal, so can multiply with[node weights |
       // bias weights] augmented matrix

    int img_is_digit = (int)lp[8 + seq];
#ifdef SAMPLEFREQ
    if ((seq + 1) % SAMPLEFREQ == 0)
    {
        cout << "For sample :" << seq + 1 << endl << flush;
        print_an_image(&mptr[start], img_is_digit);
    }
#endif
    t.zeroize();  // create the target vector (plus one for 'bias' bit)
    if (img_is_digit > 9)
    {
        cout << "Error: img_is_digit=" << img_is_digit << "seq=" << seq << endl;
        exit(1);
    }
    t.ptr[img_is_digit] = 1;    // set the target 'bit'
}


////////////////////////
//
// DEBUG ROUTINES
// For use with gdb
/*
void output(mat t)
{
     cout << t << endl;
}

// For use with gdb
void output(rowvec t)
{
     cout << t << endl;
}
*/
void output(newmat t)
{
    cout << t.prtstr();
}

double accu0(newmat m1)
{
    double tmp = 0;
    for (int i = 0; i < m1.n_rows; i++)
        for (int j = 0; j < m1.n_cols; j++)
            tmp += m1.ptr[i * m1.n_cols + j];
    return tmp;
}

#if 0
newmat diff0(newmat p1, newmat p2)
{
    newmat tmp(p1.n_rows, p1.n_cols);
    for (int i = 0; i < p1.n_rows; i++)
    {
        for (int j = 0; j < p1.n_cols; j++)
            tmp.ptr[i * p1.n_cols + j] = p1.ptr[i * p1.n_cols + j] - p2.ptr[i * p1.n_cols + j];
    }
    return tmp;
}
newmat matmult0(newmat p1, newmat p2)
{
    if (p1.n_cols == p2.n_rows)
    {
        newmat tmp(p1.n_rows, p2.n_cols);

        /*
            for(i = 0; i < r1; ++i)
                for(j = 0; j < c2; ++j)
                    for(k = 0; k < c1; ++k) // c1==r2
                    {
                        mult2[i*c2+j] += ap[i*c1+k] * bp[k*c2+j];
                    }
        */



        if (p1.n_cols == p2.n_rows)
        {
            for (int i = 0; i < p1.n_rows; i++)
                for (int j = 0; j < p2.n_cols; j++)
                    for (int k = 0; k < p1.n_cols; k++)
                        tmp.ptr[i * p2.n_cols + j] = p1.ptr[i * p1.n_cols + k] * p2.ptr[k * p2.n_cols + j];
        }
        return tmp;
    }
    else
    {
        cout << "Cant multiply p1[" << p1.n_rows << "," << p1.n_cols << "] by p2[" << p2.n_rows << "," << p2.n_cols << "]" << endl;
        exit(1);
    }
    newmat dumy;
    return dumy;
}
newmat mult(newmat p1, double p2)
{
    newmat tmp(p1.n_rows, p1.n_cols);
    for (int i = 0; i < p1.n_rows; i++)
    {
        for (int j = 0; j < p1.n_cols; j++)
            tmp.ptr[i * p1.n_cols + j] = p1.ptr[i * p1.n_cols + j] * p2;
    }
    return tmp;
}
newmat add(newmat p1, double p2)
{
    newmat tmp(p1.n_rows, p1.n_cols);
    for (int i = 0; i < p1.n_rows; i++)
    {
        for (int j = 0; j < p1.n_cols; j++)
            tmp.ptr[i * p1.n_cols + j] = p1.ptr[i * p1.n_cols + j] + p2;
    }
    return tmp;
}
newmat matadd(newmat p1, newmat p2)
{
    newmat tmp(p1.n_rows, p1.n_cols);
    for (int i = 0; i < p1.n_rows; i++)
    {
        for (int j = 0; j < p1.n_cols; j++)
            tmp.ptr[i * p1.n_cols + j] = p1.ptr[i * p1.n_cols + j] + p2.ptr[i * p1.n_cols + j];
    }
    return tmp;
}
#endif
int backprop(int y0)
{

    //tgt0.insert_cols(nodes[OutputLayer], 1);
    //  double err = accu((tgt - final) % (tgt - final)) *0.5;
    double err = 0;
    for (int i = 0; i < tgt.n_rows; i++)
        for (int j = 0; j < tgt.n_cols - 1; j++) // last ele in tgt is bias so dont include in err function
        {
            err += (tgt.ptr[i * tgt.n_cols + j] - actuation[OutputLayer].ptr[i * tgt.n_cols + j]) * (tgt.ptr[i * tgt.n_cols + j] - actuation[OutputLayer].ptr[i * tgt.n_cols + j]);
        }
    err *= 0.5;

    if (abs(err) < EPSILON)
    {
        int val = tgt.index_max_row(0, 0, 9);
#ifdef SAMPLEFREQ
        if ((y0 + 1) % SAMPLEFREQ == 0)
            cout << "---------------------------------- BACK PROPAGATION  sample=" <<
            y0 + 1 << " err=" << err << "<epsilon, for tgt '" << val <<
            "' so error is acceptable, returning" << endl << flush;
#endif
        err_summary.ptr[val] = err;
        return 1;
    }

#ifdef SAMPLEFREQ
    if ((y0 + 1) % SAMPLEFREQ == 0)
        cout << "------------------------------------ BACK PROPAGATION sample=" <<
        y0 + 1 << endl << flush;
#endif
    ftick[OutputLayer].set_mult1_add2(actuation[OutputLayer], -1.0, 1.0);                            //  ftick[OutputLayer] = -actuation[OutputLayer] + 1;
    ftick[OutputLayer].piecewisemult(actuation[OutputLayer]);	// element wise multiply                //  ftick[OutputLayer] = ftick[OutputLayer] % (actuation[OutputLayer]);      
    deltafn[OutputLayer].set_diff2_piecewisemult3(tgt, actuation[OutputLayer], ftick[OutputLayer]);  //  deltafn[OutputLayer] = (tgt0 - actuation[OutputLayer]) % (ftick[OutputLayer]);

    for (int i = OutputLayer - 1; i >= 0; i--)
    {
        deltafn_t[i + 1].set_transpose(deltafn[i + 1]);
        weight_updates[i].set_matmult(deltafn_t[i + 1], actuation[i]);            // weight_updates[i] = deltafn[i + 1].t() *actuation[i];
        new_layer_weights[i].set_mult1_add2(weight_updates[i], eta, layer_weights[i]);// new_layer_weights[i] = layer_weights[i] + (eta *weight_updates[i]);
        ftick[i].set_mult1_add2(actuation[i], -1.0, 1.0);                              //  ftick[i] = -actuation[i] + 1;
        ftick[i].piecewisemult(actuation[i]);	// element wise multiply          //  ftick[i] = ftick[i] % (actuation[i]); 
        deltafn[i].set_matmult(deltafn[i + 1], layer_weights[i]);                       // deltafn[i] = deltafn[i + 1] *layer_weights[i];
        deltafn[i].piecewisemult(ftick[i]);                                             //  deltafn[i] = deltafn[i] % ftick[i];
    }
    for (int i = 0; i < OutputLayer; i++)
    {
        for (int j = 0; j < layer_weights[i].n_rows; j++)
            for (int k = 0; k < layer_weights[i].n_cols; k++)
                layer_weights[i].ptr[j * layer_weights[i].n_cols + k] = new_layer_weights[i].ptr[j * layer_weights[i].n_cols + k];
    }
    return 0;
}


void forward_feed(unsigned char*& imgdata, unsigned char*& labdata, bool train,
    int samples)
{
    int tot_correct = 0;
    int tot_wrong = 0;
    int correct_num = -1;
    int best_guess = -1;
    int num_correct[OUTPUT_LINES] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };
    int num_wrong[OUTPUT_LINES] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };
    int chosen_wrongly[OUTPUT_LINES][OUTPUT_LINES] = {
       { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },
         { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },
         { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },
         { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },
         { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },
         { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },
         { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },
         { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },
         { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 },
         { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 }
    };
    int num_tested = 0;
    int epochs;
    if (train)
    {
        epochs = EPOCHS;
    }
    else
    {
        epochs = 1;
    }
    for (int y = 0; y < samples; y++)
    {
#ifdef SAMPLEFREQ
        if ((y + 1) % SAMPLEFREQ == 0)
        {
            cout << "------------------------------------ FORWARD FEED OF ";
            if (train)
                cout << "TRAINING";
            else cout << "TEST    ";
            cout << " SAMPLE # " << y + 1 << endl << flush;
        }
#endif
        load_an_image(y, imgdata, actuation[0], tgt, labdata);

        int tgtval = tgt.index_max_row(0, 0, 9);
        for (int e = 0; e < epochs; e++)
        {
            for (int i = 0; i < OutputLayer; i++)	// only n-1 transitions between n layers
            {
#ifdef SERIAL_ONLY
                auto StartCallTime = std::chrono::high_resolution_clock::now();


                layer_weights_t[i].set_transpose(layer_weights[i]);
                SerialMatrixVectorMultiply(netin[i].ptr,
                    actuation[i].ptr, actuation[i].n_rows,
                    layer_weights_t[i].ptr, layer_weights_t[i].n_rows, layer_weights_t[i].n_cols);
                auto EndCallTime = std::chrono::high_resolution_clock::now();
                auto TotalCallTime = std::chrono::duration_cast<std::chrono::microseconds>
                    (EndCallTime - StartCallTime);

                if (TotalCallTime > Call_MaxTime)
                    Call_MaxTime = TotalCallTime;

                if (TotalCallTime < Call_MinTime)
                    Call_MinTime = TotalCallTime;

                Avg_Time += TotalCallTime;
                avgcnt++;

                for (int j = 0; j < netin[i].n_cols; j++)
                {
                    netin[i].ptr[j] /= actuation[i].n_cols;
                }
#ifdef SAMPLEFREQ
                if ((y + 1) % SAMPLEFREQ == 0)
                    cout << "Netin serial (" << netin[i].n_rows << "," << netin[i].n_cols <<
                    ")= " << netin[i].prtstr() << endl << flush;
#endif
#else
                layer_weights_t[i].set_transpose(layer_weights[i]);
                PreMatMul(actuation[i], layer_weights_t[i], netin[i], actuation[i].n_cols);

#ifdef SAMPLEFREQ
                cout << "Netin Parallel " << netin[i].n_rows << "," << netin[i].n_cols <<
                    ")= " << netin[i].prtstr() << endl << flush;
#endif

#endif
                sigmoid3(netin[i], actuation[i + 1]);
            }
#ifdef SAMPLEFREQ
            if ((y + 1) % SAMPLEFREQ == 0)
            {
                std::cout << "Final output : " << endl << std::setw(7) << fixed <<
                    showpoint << actuation[OutputLayer].prtstr() <<
                    " Sample: " << y + 1 << std::endl << flush;
                std::cout << "Expec output : " << endl << std::setw(7) << fixed <<
                    showpoint << tgt.prtstr() << " Sample: " << y + 1 <<
                    std::endl << flush;
            }
#endif
            //////////////////////////// forward feed end
            if (train)
            {
                // printout intermediate result
           //     int outval = actuation[OutputLayer].subvec(0, 9).index_max();
                int outval = actuation[OutputLayer].index_max_row(0, 0, 9);
#ifdef SAMPLEFREQ
                if ((y + 1) % SAMPLEFREQ == 0)
                {
                    std::cout << "Train output : " << endl << std::setw(7) << fixed <<
                        showpoint << actuation[OutputLayer].prtstr() <<
                        " Sample: " << y + 1 << std::endl << flush;
                    // Below just figures out the order in which to print the "A"ctal
                    // result and "O"bjective result
                    // (or "*" if correct) in the output line.
                    // So tgtval is correct if lastval==firstval(they are indicies, and
                    // will be equal if tgtval==outval)
                    int firstval = tgtval < outval ? tgtval : outval;
                    int lastval = tgtval > outval ? tgtval : outval;
                    string firststr = tgtval == firstval ?
                        to_string(firstval) + string("T") :
                        to_string(firstval) + string("O");
                    string laststr = tgtval == lastval ? to_string(lastval) + "T" :
                        to_string(lastval) + "O";
                    if (firstval == lastval)
                        firststr = "*" + to_string(firstval);	// correct
                    for (int z1 = 0; z1 < firstval; z1++)
                        cout << "         ";
                    cout << "       " << firststr;
                    for (int z1 = 0; z1 < lastval - firstval - 1; z1++)
                        cout << "         ";
                    if (firstval != lastval)
                        cout << "       " << laststr;	// expected
                    cout << endl << flush;
                }
#endif
                if (backprop(y) == 1)
                    break;	// exit i/epoch loop and goto next sample (as error function is
               // within limits for this tgt)
            }
        }

        if (!train)
        {
            correct_num = tgt.index_max_row(0, 0, 9);
            best_guess = actuation[OutputLayer].index_max_row(0, 0, 9);

            if (best_guess == correct_num)
            {
                num_correct[correct_num]++;
                tot_correct++;
            }
            else
            {
                num_wrong[correct_num]++;
                chosen_wrongly[correct_num][best_guess]++;
                tot_wrong++;
            }
            num_tested++;
        }
        if (!train)
        {
            std::cout << "Final output : " << endl << std::setw(7) << fixed <<
                showpoint << actuation[OutputLayer].prtstr() <<
                " Sample: " << y + 1 << std::endl << flush;
            for (int z1 = 0; z1 < actuation[OutputLayer].index_max_row(0, 0, 9); z1++)
                cout << "         ";
            cout << "       ^" << endl << flush;
            std::cout << "Expec output : " << endl << std::setw(7) << fixed <<
                showpoint << tgt.prtstr() << " Sample: " << y + 1 <<
                std::endl << flush;
        }
    }
    if (!train)
    {
        confusion_matrix << "Tested         " << num_tested << " samples" << endl <<
            flush;
        confusion_matrix << "Tested Correct " << tot_correct << " samples" << endl <<
            flush;
        confusion_matrix << "Tested Wrong   " << tot_wrong << " samples" << endl <<
            endl << endl << "  " << flush;
        for (int i = 0; i < OUTPUT_LINES; i++)
            confusion_matrix << "     " << dec << std::setw(6) << "'" << i << "'";
        confusion_matrix << "<-- ANN chose" << endl << flush;
        confusion_matrix << "------------------------------------------------------"
            "------------------------------------------------------"
            "-----------------------------";
        double colsum[OUTPUT_LINES] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };
        double rowsum[OUTPUT_LINES] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };
        string blanks = "                    ";
        for (int i = 0; i < OUTPUT_LINES; i++)
        {
            string correct_size = to_string(num_correct[i]);
            confusion_matrix << endl << setw(4) << i << "  |";
            for (int j = 0; j < OUTPUT_LINES; j++)
            {
                rowsum[i] += chosen_wrongly[i][j];
                colsum[j] += chosen_wrongly[i][j];
                if (i == j)
                    confusion_matrix << std::setw(6) << "[" << num_correct[i] << "]" <<
                    blanks.substr(0, 5 - correct_size.length()) <<
                    "|";
                else
                    confusion_matrix << std::setw(7) << chosen_wrongly[i][j] << "     |";
            }
            float pctg = 0;
            if (tot_wrong != 0)
                pctg = (float)(rowsum[i]) / (float)(tot_wrong) * 100.0f;
            confusion_matrix << "  " << setw(7) << std::setw(7);
            confusion_matrix.copyfmt(init);
            confusion_matrix << rowsum[i];
            confusion_matrix << setw(7) << "     " << fixed << showpoint << pctg <<
                "%" << endl << flush;
            confusion_matrix.copyfmt(init);
            confusion_matrix << "----------------------------------------------------"
                "----------------------------------------------------"
                "---------------------------------";
        }
        confusion_matrix << endl << "   ^   ";
        for (int i = 0; i < OUTPUT_LINES; i++)
            confusion_matrix << dec << std::setw(7) << colsum[i] << "      ";
        confusion_matrix << endl << "Target   ";
        for (int i = 0; i < OUTPUT_LINES; i++)
        {
            float pctg = 0;
            if (tot_wrong != 0)
                pctg = (float)(colsum[i]) / (float)(tot_wrong) * 100.0f;
            confusion_matrix << dec << setw(7) << fixed << showpoint << pctg <<
                "%     ";
            confusion_matrix.copyfmt(init);
        }
        confusion_matrix << endl << endl << endl << endl << endl <<
            "Correct selections:" << endl << flush;
        confusion_matrix << "       ";
        for (int i = 0; i < OUTPUT_LINES; i++)
            confusion_matrix << dec << std::setw(6) << "'" << i << "'     ";
        confusion_matrix << endl << "       ";
        for (int i = 0; i < OUTPUT_LINES; i++)
        {
            confusion_matrix << std::setw(7) << num_correct[i] << "      ";
        }
        confusion_matrix << endl << endl << "Incorrect selections:" << endl <<
            flush;
        confusion_matrix << "       ";
        for (int i = 0; i < OUTPUT_LINES; i++)
            confusion_matrix << dec << std::setw(6) << "'" << i << "'     ";
        confusion_matrix << endl << "       ";
        for (int i = 0; i < OUTPUT_LINES; i++)
        {
            confusion_matrix << std::setw(7) << num_wrong[i] << "      ";
        }
        confusion_matrix << endl << endl << flush;
        float pctg =
            (float)(tot_correct) / (float)(tot_correct + tot_wrong) * 100.0f;
        confusion_matrix << "Total Correct : " << std::setw(7) << fixed << showpoint <<
            pctg << "%     " << endl << endl << flush;
        cout << confusion_matrix.str() << flush;
        confusion_matrix.copyfmt(init);
        cout.copyfmt(init);
    }
}

#ifdef WANT_TO_LOAD_WEIGHTS
void load_weights(string fname)
{
    ifstream iFile;
    cout << "Loading weights from file : " << fname << endl << flush;
    iFile.open(fname, ios::in);
    string aline;
    int olaycnt = 0;

    vector<string> strs;

    if (fname.substr(0, 4) == "post")
        stringstream confusion_matrix2;
    getline(iFile, aline);
    boost::split(strs, aline, boost::is_any_of("="));
    if (strs.size() > 1)
        lays = stoi(strs[1]);
    cout << "Layer # " << ++olaycnt << " Has " << lays << " nodes" << endl;
    while (iFile.good())
    {
        getline(iFile, aline);
        if (aline.find("NodesInLayer") != std::string::npos)
        {
            nd2[t] = x;
            x = 0;
            t++;

            boost::split(strs, aline, boost::is_any_of("="));
            if (strs.size() > 1)
                nd[t] = stoi(strs[1]);
            cout << "Layer # " << ++olaycnt << " Has " << nd[t] << "layers" << endl;
        }
        else if ((aline.find("Error Summary") != std::string::npos))
        {
            nd2[t] = x;
            x = 0;
            t++;
            break;
        }
        else if (aline.find("LayerBiases") == std::string::npos)
        {
            boost::trim(aline);
            boost::split(strs, aline, boost::is_any_of(" "));
            boost::algorithm::split(strs, aline, boost::is_any_of("\t "),
                boost::token_compress_on);
            for (int y = 0; y < strs.size(); y++)
            {
                if (strs[y].length() > 0)
                {
                    l2[t][x++] = stod(strs[y]);
                }
            }
        }
    }
    for (int i = 0; i < lays; i++)
    {
        for (int j = 0; j < nd2[i + 1]; j++)
        {
            int r = (j) / (nd[i + 1] + 1);
            int c = (j) % (nd[i + 1] + 1);
            layer_weights[i].ptr[r * (nd[i + 1] + 1) + c] = l2[i + 1][j];
        }
    }
}
#endif

void save_weights(string hdr)
{
    ofstream oFile;
    string fname = hdr + string("_weights_") + fid + string(".txt");
    cout << "Saving weights to file : " << fname << endl << flush;
    oFile.open(fname, ios::out);
    if (hdr.substr(0, 4) == "post")
        oFile << confusion_matrix.str();
    oFile << "NumberOfLayers=" << NumberOfLayers << endl << flush;
    for (int i = 0; i < OutputLayer; i++)
    {

        oFile << "NodesInLayer" << i << "=" << nodes[i] << endl << setprecision(20) << fixed << showpoint << flush;
        oFile << "    " << layer_weights[i].prtstr();
        oFile << endl;
        // ALTERNATIVELY
        //for (int j=0; j<layer_weights[i].n_rows ;j++)
        //{
        //   for (int k=0; k<layer_weights[i].n_cols; k++)
        //        oFile << setprecision(20) <<  fixed << showpoint << layer_weights[i](j,k) << " " <<flush;
        //    oFile << endl;
        // }
    }
    oFile << "Error Summary" << endl << flush;

    oFile << err_summary.prtstr() << endl << flush;

    oFile << "EndFile" << endl << flush;
    oFile.close();
    cout.copyfmt(init);

}

int main()
{

    size_t available, total;
    hipMemGetInfo(&available, &total);
    cout << "avail=" << available << " total=" << total << endl;
    int nDevices;

    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (KHz): %d\n",
            prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",
            prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
            2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
    }
    extern char** environ;
    string hname = "";
    //string y="initial_random_values_weights_11337071.txt";
    string weight_file_to_preload = "initial_random_values_weights_1637223695.txt";
    auto StartChronoTime = std::chrono::high_resolution_clock::now();

    vector<string> strs;
    string bldver = string(__DATE__) + " at time " + string(__TIME__);
    cout << "--------------------------------  Build done on " << bldver << endl <<
        flush;
    init.copyfmt(cout);
    for (int i = 0; i < err_summary.n_cols; i++)
        err_summary.ptr[i] = -1.0;
   
        NumberOfLayers = 3;
        nodes = new unsigned int[NumberOfLayers];
        nodes[0] = INPUT_LINES;
        nodes[1] = DEFAULT_HIDDEN;
        nodes[2] = OUTPUT_LINES;
        eta = ETA_DEFAULT;
        cout << "Using default setting of \"" << nodes[0] << " " << nodes[1] << " " <<
            nodes[2] << "\" " << endl << flush;
        cout << "And ETA=" << eta << endl << flush;;
    



    cout << "Number of Layers is " << NumberOfLayers << endl << flush;


#ifndef SERIAL_ONLY
         // set up CUDA timing structs
    hipEventCreate(&start);
    hipEventCreate(&stop);
#endif

    OutputLayer = NumberOfLayers - 1;
    unsigned char* trainlabels;
    unsigned char* testlabels;
    unsigned char* traindata = load_file("train-images-idx3-ubyte",
        "train-labels-idx1-ubyte", &trainlabels);
    unsigned char* testdata = load_file("t10k-images-idx3-ubyte",
        "t10k-labels-idx1-ubyte", &testlabels);
    auto StartTime = std::chrono::high_resolution_clock::now();

    ///////////////////////////////////////////////
    //
    //  CREATE ARRAY OF MATRICES AND VECTORS
    //  AND SET WEIGHTS TO RANDOM (0<w < 1)
    //
    int max_mat = 0;
    int max_vec = 0;
    int bias_field = 1;

    for (int i = 0; i <= OutputLayer; i++)
    {
        max_vec = max(max_vec, (nodes[i] + bias_field));
        newmat rb3a(1, nodes[i] + bias_field);
        actuation.push_back(rb3a);

        newmat drb3(1, nodes[i] + bias_field);
        deltafn.push_back(drb3);

        newmat drb3_t(nodes[i] + bias_field, 1);
        deltafn_t.push_back(drb3_t);

        newmat frb3(1, nodes[i] + bias_field);
        ftick.push_back(frb3);

        if (i < OutputLayer)
        {
            max_mat =
                max(max_mat, (nodes[i] + bias_field) * (nodes[i + 1] + bias_field));
            // These buffers for the rowvec and mat structures below are done to ensure
            // the Armadillo matrix can be accessed directly and the library doesnt move
            // the memory around
            newmat rb3(1, (nodes[i + 1] + bias_field));

            // Create an array of matrices (one element for each layer) for the netin value
            // This holds the sum of weighted signals, for each node, that gets squashed to 
            // produce the nodes output for next layer
            netin.push_back(rb3);
            // Create a buffer of required size for weights, in each layer
            // (plus two more, one for delta updates, and one for holding new weight to be
            // applied after backprop. These maybe consolidated later
            newmat tmpwgt3((nodes[i + 1] + bias_field), (nodes[i] + bias_field));
            newmat tmpwgt3_t((nodes[i] + bias_field), (nodes[i + 1] + bias_field));
            for (int p = 0; p < (nodes[i + 1] + bias_field) * (nodes[i] + bias_field); p++)
                tmpwgt3.ptr[p] = (double)rand() / (double)RAND_MAX;
            newmat tmpwgt30((nodes[i + 1] + bias_field), (nodes[i] + bias_field));
            newmat tmpwgt300((nodes[i + 1] + bias_field), (nodes[i] + bias_field));
            // create an array of three matrices (weights for forward prop)
            // and deltas and new values, for back propagation
            layer_weights.push_back(tmpwgt3);
            layer_weights_t.push_back(tmpwgt3_t);

            new_layer_weights.push_back(tmpwgt30);

            weight_updates.push_back(tmpwgt300);
        }
    }

    // Informational, the max value of matrix and vectors are record and used to reserve CUDA memory 
    cout << "Max Matrix size " << max_mat << " Max vector size = " << max_vec <<
        endl << flush;

#ifdef WANT_TO_LOAD_WEIGHTS
    // this is a function to load previously saved weights, to either ensure constant initial values
    // if say moving platforms with different psudeo RNG, or to load post weights after training
    // This works, but only implemented atm, by direct code changes, no UI implemented
    // But note used in this project anyway
    cout << "Chosen to load saved weight file '" << weight_file_to_preload << "' , so loading it ....." << endl;
    load_weights(weight_file_to_preload);
#else
    // Save initial starting weights if required for later
    save_weights("initial_random_values");
#endif

#ifndef SERIAL_ONLY
    checkError(hipMalloc((void**)&deviceA, max_mat * sizeof(double)));
    checkError(hipMalloc((void**)&deviceB, max_mat * sizeof(double)));
    checkError(hipMalloc((void**)&deviceC, max_mat * sizeof(double)));

#ifdef __CUDA_ARCH__
    cout << "Built for CUDA ARCH == " << __CUDA_ARCH__ << endl;
#endif
#endif
    ///////////////////////////////////////////////
    //
    // TRAIN THE DATA
    //
    auto StartTrainTime = std::chrono::high_resolution_clock::now();
    cout << "Training on data started (epochs=" << EPOCHS << ")...." << endl <<
        flush;

    forward_feed(traindata, trainlabels, true, TRAININGSAMPLES);
    auto EndTrainTime = std::chrono::high_resolution_clock::now();

    cout << "Training complete" << endl << flush;
    ///////////////////////////////////////////////
    //
    // TEST THE DATA
    //
    cout << "Testing of data started...." << endl << flush;
    auto StartTestTime = std::chrono::high_resolution_clock::now();

    forward_feed(testdata, testlabels, false, TESTINGSAMPLES);

    auto EndTestTime = std::chrono::high_resolution_clock::now();

    cout << "Testing complete" << endl << flush;

    auto TotalTime = std::chrono::duration_cast<std::chrono::microseconds> (EndTestTime - StartTime);
    auto TrainTime = std::chrono::duration_cast<std::chrono::microseconds> (EndTrainTime - StartTrainTime);
    auto TestTime = std::chrono::duration_cast<std::chrono::microseconds> (EndTestTime - StartTestTime);

    cout << "Total Time       : " << std::setw(12) << TotalTime.count() << " us" <<
        endl << flush;
    cout << "Total Train Time : " << std::setw(12) << TrainTime.count() << " us" <<
        endl << flush;
    cout << "Total Test Time  : " << std::setw(12) << TestTime.count() << " us" <<
        endl << flush;

    confusion_matrix << "Epochs in Training : " << EPOCHS << endl << flush;
    confusion_matrix << "Training Samples   : " << TRAININGSAMPLES << endl <<
        flush;
    confusion_matrix << "Testing Samples    : " << TESTINGSAMPLES << endl <<
        flush;
    confusion_matrix << endl << endl << "Total Time       : " << std::setw(12) <<
        TotalTime.count() << " us" << endl << flush;
    confusion_matrix << "Total Train Time : " << std::setw(12) <<
        TrainTime.count() << " us" << endl << flush;
    confusion_matrix << "Total Test Time  : " << std::setw(12) << TestTime.count() <<
        " us" << endl << flush;
    confusion_matrix << endl << endl << "Total Time       : " << std::setw(12) <<
        TotalTime.count() / 1000000 << " s" << endl << flush;
    confusion_matrix << "Total Train Time : " << std::setw(12) <<
        TrainTime.count() / 1000000 << " s" << endl << flush;
    confusion_matrix << "Total Test Time  : " << std::setw(12) <<
        TestTime.count() / 1000000 << " s" << endl << flush;
    confusion_matrix << endl << endl << "Total Time       : " << std::setw(12) <<
        TotalTime.count() / 60000000 << " min" << endl << flush;
    confusion_matrix << "Total Train Time : " << std::setw(12) <<
        TrainTime.count() / 60000000 << " min" << endl << flush;
    confusion_matrix << "Total Test Time  : " << std::setw(12) <<
        TestTime.count() / 60000000 << " min" << endl << flush;
    confusion_matrix << "Epsilon  : " << EPSILON << endl << flush;
    confusion_matrix << "Eta      : " << eta << endl << flush;
    confusion_matrix << "Build ver: " << bldver << endl << flush;
    save_weights("post_training_weights");

    delete[] traindata;
    delete[] trainlabels;
    delete[] testdata;
    delete[] testlabels;

    cout << "Min time for " << build_type << " call : " << Call_MinTime.count() << " us" << endl;
    cout << "Min time for " << build_type << " call : " << Call_MinTime.count() / 1000000 << " s" << endl;
    cout << "Min time for " << build_type << " call : " << Call_MinTime.count() / 60000000 << " min" << endl;
    cout << "Avg time for " << build_type << " call : " << (double)Avg_Time.count() / (double)avgcnt << " us" << endl;
    cout << "Avg time for " << build_type << " call : " << (double)Avg_Time.count() / (double)(avgcnt * 1000000) << " s" << endl;
    cout << "Avg time for " << build_type << " call : " << (double)Avg_Time.count() / (double)(avgcnt * 60000000) << " min" << endl;
    cout << "Max time for " << build_type << " call : " << Call_MaxTime.count() << " us" << endl;
    cout << "Max time for " << build_type << " call : " << Call_MaxTime.count() / 1000000 << " s" << endl;
    cout << "Max time for " << build_type << " call : " << Call_MaxTime.count() / 60000000 << " min" << endl;
    auto EndChronoTime = std::chrono::high_resolution_clock::now();
    auto TotalChronoTime = std::chrono::duration_cast<std::chrono::microseconds> (EndChronoTime - StartChronoTime);

    cout << "Time for  Total Program : " << TotalChronoTime.count() << " us " << endl;

    for (int i = 0; i < netin.size(); i++)
        netin[i].free_ele();


#ifndef SERIAL_ONLY
    cout << "Used Tile Dimension of " << tile_dimension << endl;
    checkError(hipFree(deviceA));
    checkError(hipFree(deviceB));
    checkError(hipFree(deviceC));

    checkError(hipEventDestroy(start));
    checkError(hipEventDestroy(stop));
#endif


}


